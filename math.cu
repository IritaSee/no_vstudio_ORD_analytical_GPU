
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void multiply(float *a, float *b, float *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] * b[i];
  }
}


int main(){
   
    float A[5] = { 1, 2, 3, 4, 5 };
    float B[5] = { 10, 20, 30, 40, 50 };
    float C[5];
    multiply<<<1,5>>>(A, B, C, 5);
    int i;
    for(i=0;i<5;i++){
        printf("%f\n",C[i]);
    }
    return 0;
}