#include "hip/hip_runtime.h"

#include <iostream>

using namespace std;

double devAr[4];
double *d_devAr;
__global__ void foo(double *devAr)
{
    // printf("\n");
    // printf("dev before init: %d \n", devAr[0]);
    devAr[0] = 0.77;
    // printf("dev after init: %d \n", devAr[0]);
    // printf("\n");
}

int main()
{
    double test[4];
    devAr[0] = 0.0;
    devAr[1] = 0.33;
    devAr[2] = 0.66;
    devAr[3] = 0.99;
    cout << "host initially: " << devAr[0] << endl;
    devAr[0] = 7.0;
    cout << "host after init: " << devAr[0] << endl;
    printf("malloc: %d\n",hipMalloc((void**)&d_devAr, 4*sizeof(double))); 
    // cudaMalloc((void**)&d_devAr, 5*sizeof(double)); 
    printf("\nstatus: %d\n",hipMemcpy(d_devAr, devAr, 4*sizeof(double), hipMemcpyHostToDevice));
    //foo << <1, 1 >> >(*d_devAr);
    hipDeviceSynchronize();
    printf("\nstatus: %d\n",hipMemcpy(test, d_devAr, 4*sizeof(double), hipMemcpyDeviceToHost));
    for (int a=0; a<4;a++){
        cout << "after foo: " << test[a] << endl;
    }
    
}