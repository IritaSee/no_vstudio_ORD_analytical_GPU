
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(float *a, float *b, float *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("%d\n",i);
  if (i < n) {
    c[i] = a[i] * b[i];
  }
}

int main() {
  float A[6] = { 1, 2, 3, 4, 5, 6 };
  float B[6] = { 10, 20, 30, 40, 50, 60 };
  float C[6] = { 0 };
  multiply<<<1,6>>>(A, B, C, 6);
  hipDeviceSynchronize();
  int i;
  for(i=0;i<6;i++){
    printf("%f * %f = %f\n",A[i],B[i],C[i]);
  }
  return 0;
}