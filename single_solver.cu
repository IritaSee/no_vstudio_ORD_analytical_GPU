#include "hip/hip_runtime.h"
// CVodeSimTestSimple.cpp : This file contains the 'main' function. Program execution begins and ends there.
//
#include <array>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstring>
#include <ctime>
#include <iostream>

#include <hip/hip_runtime.h>

#include "enums/enum_mar_cell_MKII.cuh"

#include "modules/globals.hpp"
#include "modules/commons.hpp"


clock_t START_TIMER;

char buffer[255];
double concs [4] = {0.0, 33.0, 66.0, 99.0};
// variables for I/O
FILE* fp_vm;
FILE* fp_gate;
    
// timer section
clock_t tic()
/* start timer*/
{
    return START_TIMER = clock();
}

void toc(clock_t start)
{
  /*stop timer*/
    std::cout
        << "Elapsed time: "
        << (clock() - start) / (double)CLOCKS_PER_SEC << "s"
        << std::endl;
}


void get_IC50_data_from_file(const char* file_name);

__global__ void initConsts(double* CONSTANTS, double* RATES, double *STATES){
CONSTANTS[celltype] = 0;
CONSTANTS[R] = 8314;
CONSTANTS[T] = 310;
CONSTANTS[F] = 96485;
CONSTANTS[cm] = 1;
CONSTANTS[rad] = 0.0011;
CONSTANTS[L] = 0.01;
CONSTANTS[vcell] =  1000.00*3.14000*CONSTANTS[rad]*CONSTANTS[rad]*CONSTANTS[L];
CONSTANTS[amp] = -80;
CONSTANTS[duration] = 0.5;
CONSTANTS[zna] = 1;
CONSTANTS[zca] = 2;
CONSTANTS[zk] = 1;
CONSTANTS[stim_start] = 10.0;
CONSTANTS[stim_end] = 100000000000000000;
CONSTANTS[stim_period] = 1000.0;
CONSTANTS[step_low] = -150.;
CONSTANTS[step_high] = 0;
CONSTANTS[step_start] = 10;
CONSTANTS[step_end] = 5000;
CONSTANTS[GNa] = 75;
CONSTANTS[CaMKo] = 0.05;
CONSTANTS[KmCaM] = 0.0015;
CONSTANTS[KmCaMK] = 0.15;
CONSTANTS[nao] = 140;
CONSTANTS[mssV1] = 39.57;
CONSTANTS[mssV2] = 9.871;
CONSTANTS[mtD1] = 6.765;
CONSTANTS[mtD2] = 8.552;
CONSTANTS[mtV1] = 11.64;
CONSTANTS[mtV2] = 34.77;
CONSTANTS[mtV3] = 77.42;
CONSTANTS[mtV4] = 5.955;
CONSTANTS[hssV1] = 82.9;
CONSTANTS[hssV2] = 6.086;
CONSTANTS[Ahf] = 0.99;
CONSTANTS[Ahs] = 1.00000 - CONSTANTS[Ahf];
CONSTANTS[GNaL_b] = 0.0075;
CONSTANTS[GNaL] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[GNaL_b]*0.600000 : CONSTANTS[GNaL_b]);
CONSTANTS[thL] = 200;
CONSTANTS[thLp] =  3.00000*CONSTANTS[thL];
CONSTANTS[PNab] = 3.75e-10;
CONSTANTS[Gto_b] = 0.02;
CONSTANTS[Gto] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[Gto_b]*4.00000 : CONSTANTS[celltype]==2.00000 ?  CONSTANTS[Gto_b]*4.00000 : CONSTANTS[Gto_b]);
CONSTANTS[ko] = 5.4;
CONSTANTS[GKr_b] = 0.046;
CONSTANTS[GKr] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[GKr_b]*1.30000 : CONSTANTS[celltype]==2.00000 ?  CONSTANTS[GKr_b]*0.800000 : CONSTANTS[GKr_b]);
CONSTANTS[GKs_b] = 0.0034;
CONSTANTS[GKs] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[GKs_b]*1.40000 : CONSTANTS[GKs_b]);
CONSTANTS[PKNa] = 0.01833;
CONSTANTS[GK1_b] = 0.1908;
CONSTANTS[GK1] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[GK1_b]*1.20000 : CONSTANTS[celltype]==2.00000 ?  CONSTANTS[GK1_b]*1.30000 : CONSTANTS[GK1_b]);
CONSTANTS[GKb_b] = 0.003;
CONSTANTS[GKb] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[GKb_b]*0.600000 : CONSTANTS[GKb_b]);
CONSTANTS[Kmn] = 0.002;
CONSTANTS[k2n] = 1000;
CONSTANTS[tjca] = 75.0000;
CONSTANTS[Aff] = 0.600000;
CONSTANTS[Afs] = 1.00000 - CONSTANTS[Aff];
CONSTANTS[PCa_b] = 0.0001;
CONSTANTS[PCa] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[PCa_b]*1.20000 : CONSTANTS[celltype]==2.00000 ?  CONSTANTS[PCa_b]*2.50000 : CONSTANTS[PCa_b]);
CONSTANTS[PCaK] =  0.000357400*CONSTANTS[PCa];
CONSTANTS[PCaNa] =  0.00125000*CONSTANTS[PCa];
CONSTANTS[PCap] =  1.10000*CONSTANTS[PCa];
CONSTANTS[PCaKp] =  0.000357400*CONSTANTS[PCap];
CONSTANTS[PCaNap] =  0.00125000*CONSTANTS[PCap];
CONSTANTS[cao] = 1.8;
CONSTANTS[PCab] = 2.5e-8;
CONSTANTS[GpCa] = 0.0005;
CONSTANTS[KmCap] = 0.0005;
CONSTANTS[kasymm] = 12.5;
CONSTANTS[kcaon] = 1.5e6;
CONSTANTS[kcaoff] = 5e3;
CONSTANTS[kna1] = 15;
CONSTANTS[kna2] = 5;
CONSTANTS[kna3] = 88.12;
CONSTANTS[qna] = 0.5224;
CONSTANTS[qca] = 0.167;
CONSTANTS[wnaca] = 5e3;
CONSTANTS[wna] = 6e4;
CONSTANTS[wca] = 6e4;
CONSTANTS[KmCaAct] = 150e-6;
CONSTANTS[Gncx_b] = 0.0008;
CONSTANTS[Gncx] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[Gncx_b]*1.10000 : CONSTANTS[celltype]==2.00000 ?  CONSTANTS[Gncx_b]*1.40000 : CONSTANTS[Gncx_b]);
CONSTANTS[h10_i] = CONSTANTS[kasymm]+1.00000+ (CONSTANTS[nao]/CONSTANTS[kna1])*(1.00000+CONSTANTS[nao]/CONSTANTS[kna2]);
CONSTANTS[h11_i] = ( CONSTANTS[nao]*CONSTANTS[nao])/( CONSTANTS[h10_i]*CONSTANTS[kna1]*CONSTANTS[kna2]);
CONSTANTS[h12_i] = 1.00000/CONSTANTS[h10_i];
CONSTANTS[k1_i] =  CONSTANTS[h12_i]*CONSTANTS[cao]*CONSTANTS[kcaon];
CONSTANTS[k2_i] = CONSTANTS[kcaoff];
CONSTANTS[k5_i] = CONSTANTS[kcaoff];
CONSTANTS[h10_ss] = CONSTANTS[kasymm]+1.00000+ (CONSTANTS[nao]/CONSTANTS[kna1])*(1.00000+CONSTANTS[nao]/CONSTANTS[kna2]);
CONSTANTS[h11_ss] = ( CONSTANTS[nao]*CONSTANTS[nao])/( CONSTANTS[h10_ss]*CONSTANTS[kna1]*CONSTANTS[kna2]);
CONSTANTS[h12_ss] = 1.00000/CONSTANTS[h10_ss];
CONSTANTS[k1_ss] =  CONSTANTS[h12_ss]*CONSTANTS[cao]*CONSTANTS[kcaon];
CONSTANTS[k2_ss] = CONSTANTS[kcaoff];
CONSTANTS[k5_ss] = CONSTANTS[kcaoff];
CONSTANTS[k1p] = 949.5;
CONSTANTS[k2p] = 687.2;
CONSTANTS[k3p] = 1899;
CONSTANTS[k4p] = 639;
CONSTANTS[k1m] = 182.4;
CONSTANTS[k2m] = 39.4;
CONSTANTS[k3m] = 79300;
CONSTANTS[k4m] = 40;
CONSTANTS[Knai0] = 9.073;
CONSTANTS[Knao0] = 27.78;
CONSTANTS[delta] = -0.155;
CONSTANTS[Kki] = 0.5;
CONSTANTS[Kko] = 0.3582;
CONSTANTS[MgADP] = 0.05;
CONSTANTS[MgATP] = 9.8;
CONSTANTS[H] = 1e-7;
CONSTANTS[Kmgatp] = 1.698e-7;
CONSTANTS[eP] = 4.2;
CONSTANTS[Khp] = 1.698e-7;
CONSTANTS[Knap] = 224;
CONSTANTS[Kxkur] = 292;
CONSTANTS[a2] = CONSTANTS[k2p];
CONSTANTS[a4] = (( CONSTANTS[k4p]*CONSTANTS[MgATP])/CONSTANTS[Kmgatp])/(1.00000+CONSTANTS[MgATP]/CONSTANTS[Kmgatp]);
CONSTANTS[b1] =  CONSTANTS[k1m]*CONSTANTS[MgADP];
CONSTANTS[Pnak_b] = 30;
CONSTANTS[Pnak] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[Pnak_b]*0.900000 : CONSTANTS[celltype]==2.00000 ?  CONSTANTS[Pnak_b]*0.700000 : CONSTANTS[Pnak_b]);
CONSTANTS[upScale] = (CONSTANTS[celltype]==1.00000 ? 1.30000 : 1.00000);
CONSTANTS[bt] = 4.75;
CONSTANTS[btp] =  1.25000*CONSTANTS[bt];
CONSTANTS[a_relp] =  0.500000*CONSTANTS[btp];
CONSTANTS[a_rel] =  0.500000*CONSTANTS[bt];
CONSTANTS[aCaMK] = 0.05;
CONSTANTS[bCaMK] = 0.00068;
CONSTANTS[Ageo] =  2.00000*3.14000*CONSTANTS[rad]*CONSTANTS[rad]+ 2.00000*3.14000*CONSTANTS[rad]*CONSTANTS[L];
CONSTANTS[Acap] =  2.00000*CONSTANTS[Ageo];
CONSTANTS[vmyo] =  0.680000*CONSTANTS[vcell];
CONSTANTS[vss] =  0.0200000*CONSTANTS[vcell];
CONSTANTS[vjsr] =  0.00480000*CONSTANTS[vcell];
CONSTANTS[vnsr] =  0.0552000*CONSTANTS[vcell];
CONSTANTS[cmdnmax_b] = 0.05;
CONSTANTS[cmdnmax] = (CONSTANTS[celltype]==1.00000 ?  CONSTANTS[cmdnmax_b]*1.30000 : CONSTANTS[cmdnmax_b]);
CONSTANTS[kmcmdn] = 0.00238;
CONSTANTS[trpnmax] = 0.07;
CONSTANTS[kmtrpn] = 0.0005;
CONSTANTS[BSLmax] = 1.124;
CONSTANTS[BSRmax] = 0.047;
CONSTANTS[KmBSR] = 0.00087;
CONSTANTS[KmBSL] = 0.0087;
CONSTANTS[csqnmax] = 10;
CONSTANTS[kmcsqn] = 0.8;
STATES[m] = 0;
STATES[j] = 1;
STATES[jp] = 1;
STATES[hf] = 1;
STATES[hs] = 1;
STATES[hsp] = 1;
STATES[V] = -87;
STATES[CaMKt] = 0;
STATES[cass] = 1e-4;
STATES[nai] = 7;
STATES[mL] = 0;
STATES[hL] = 1;
STATES[hLp] = 1;
STATES[a] = 0;
STATES[ap] = 0;
STATES[ki] = 145;
STATES[iF] = 1;
STATES[iS] = 1;
STATES[iFp] = 1;
STATES[iSp] = 1;
STATES[xrf] = 0;
STATES[xrs] = 0;
STATES[xs1] = 0;
STATES[xs2] = 0;
STATES[cai] = 1e-4;
STATES[xk1] = 1;
STATES[d] = 0;
STATES[ff] = 1;
STATES[fs] = 1;
STATES[fcaf] = 1;
STATES[nca] = 0;
STATES[jca] = 1;
STATES[fcas] = 1;
STATES[ffp] = 1;
STATES[fcafp] = 1;
STATES[kss] = 145;
STATES[nass] = 7;
STATES[cansr] = 1.2;
STATES[Jrelnp] = 0;
STATES[Jrelp] = 0;
STATES[cajsr] = 1.2;
}

__global__ void computeRates(double TIME, double* CONSTANTS, double* RATES, double* STATES, double* ALGEBRAIC)
{
ALGEBRAIC[vffrt] = ( STATES[V]*CONSTANTS[F]*CONSTANTS[F])/( CONSTANTS[R]*CONSTANTS[T]);
ALGEBRAIC[vfrt] = ( STATES[V]*CONSTANTS[F])/( CONSTANTS[R]*CONSTANTS[T]);
ALGEBRAIC[Istim] = (TIME>=CONSTANTS[stim_start]&&TIME<=CONSTANTS[stim_end]&&(TIME - CONSTANTS[stim_start]) -  floor((TIME - CONSTANTS[stim_start])/CONSTANTS[stim_period])*CONSTANTS[stim_period]<=CONSTANTS[duration] ? CONSTANTS[amp] : 0.00000);
ALGEBRAIC[mss] = 1.00000/(1.00000+exp(- (STATES[V]+CONSTANTS[mssV1])/CONSTANTS[mssV2]));
ALGEBRAIC[tm] = 1.00000/( CONSTANTS[mtD1]*exp((STATES[V]+CONSTANTS[mtV1])/CONSTANTS[mtV2])+ CONSTANTS[mtD2]*exp(- (STATES[V]+CONSTANTS[mtV3])/CONSTANTS[mtV4]));
ALGEBRAIC[hss] = 1.00000/(1.00000+exp((STATES[V]+CONSTANTS[hssV1])/CONSTANTS[hssV2]));
ALGEBRAIC[ths] = 1.00000/( 0.00979400*exp(- (STATES[V]+17.9500)/28.0500)+ 0.334300*exp((STATES[V]+5.73000)/56.6600));
ALGEBRAIC[thf] = 1.00000/( 1.43200e-05*exp(- (STATES[V]+1.19600)/6.28500)+ 6.14900*exp((STATES[V]+0.509600)/20.2700));
ALGEBRAIC[h] =  CONSTANTS[Ahf]*STATES[hf]+ CONSTANTS[Ahs]*STATES[hs];
ALGEBRAIC[jss] = ALGEBRAIC[hss];
ALGEBRAIC[tj] = 2.03800+1.00000/( 0.0213600*exp(- (STATES[V]+100.600)/8.28100)+ 0.305200*exp((STATES[V]+0.994100)/38.4500));
ALGEBRAIC[hssp] = 1.00000/(1.00000+exp((STATES[V]+89.1000)/6.08600));
ALGEBRAIC[thsp] =  3.00000*ALGEBRAIC[ths];
ALGEBRAIC[hp] =  CONSTANTS[Ahf]*STATES[hf]+ CONSTANTS[Ahs]*STATES[hsp];
ALGEBRAIC[tjp] =  1.46000*ALGEBRAIC[tj];
ALGEBRAIC[ENa] =  (( CONSTANTS[R]*CONSTANTS[T])/CONSTANTS[F])*log(CONSTANTS[nao]/STATES[nai]);
ALGEBRAIC[CaMKb] = ( CONSTANTS[CaMKo]*(1.00000 - STATES[CaMKt]))/(1.00000+CONSTANTS[KmCaM]/STATES[cass]);
ALGEBRAIC[CaMKa] = ALGEBRAIC[CaMKb]+STATES[CaMKt];
ALGEBRAIC[fINap] = 1.00000/(1.00000+CONSTANTS[KmCaMK]/ALGEBRAIC[CaMKa]);
ALGEBRAIC[INa] =  CONSTANTS[GNa]*(STATES[V] - ALGEBRAIC[ENa])*pow(STATES[m], 3.00000)*( (1.00000 - ALGEBRAIC[fINap])*ALGEBRAIC[h]*STATES[j]+ ALGEBRAIC[fINap]*ALGEBRAIC[hp]*STATES[jp]);
ALGEBRAIC[mLss] = 1.00000/(1.00000+exp(- (STATES[V]+42.8500)/5.26400));
ALGEBRAIC[tmL] = ALGEBRAIC[tm];
ALGEBRAIC[hLss] = 1.00000/(1.00000+exp((STATES[V]+87.6100)/7.48800));
ALGEBRAIC[hLssp] = 1.00000/(1.00000+exp((STATES[V]+93.8100)/7.48800));
ALGEBRAIC[fINaLp] = 1.00000/(1.00000+CONSTANTS[KmCaMK]/ALGEBRAIC[CaMKa]);
ALGEBRAIC[INaL] =  CONSTANTS[GNaL]*(STATES[V] - ALGEBRAIC[ENa])*STATES[mL]*( (1.00000 - ALGEBRAIC[fINaLp])*STATES[hL]+ ALGEBRAIC[fINaLp]*STATES[hLp]);
ALGEBRAIC[INab] = ( CONSTANTS[PNab]*ALGEBRAIC[vffrt]*( STATES[nai]*exp(ALGEBRAIC[vfrt]) - CONSTANTS[nao]))/(exp(ALGEBRAIC[vfrt]) - 1.00000);
ALGEBRAIC[ass] = 1.00000/(1.00000+exp(- (STATES[V] - 14.3400)/14.8200));
ALGEBRAIC[ta] = 1.05150/(1.00000/( 1.20890*(1.00000+exp(- (STATES[V] - 18.4099)/29.3814)))+3.50000/(1.00000+exp((STATES[V]+100.000)/29.3814)));
ALGEBRAIC[iss] = 1.00000/(1.00000+exp((STATES[V]+43.9400)/5.71100));
ALGEBRAIC[delta_epi] = (CONSTANTS[celltype]==1.00000 ? 1.00000 - 0.950000/(1.00000+exp((STATES[V]+70.0000)/5.00000)) : 1.00000);
ALGEBRAIC[tiF_b] = 4.56200+1.00000/( 0.393300*exp(- (STATES[V]+100.000)/100.000)+ 0.0800400*exp((STATES[V]+50.0000)/16.5900));
ALGEBRAIC[tiS_b] = 23.6200+1.00000/( 0.00141600*exp(- (STATES[V]+96.5200)/59.0500)+ 1.78000e-08*exp((STATES[V]+114.100)/8.07900));
ALGEBRAIC[tiF] =  ALGEBRAIC[tiF_b]*ALGEBRAIC[delta_epi];
ALGEBRAIC[tiS] =  ALGEBRAIC[tiS_b]*ALGEBRAIC[delta_epi];
ALGEBRAIC[AiF] = 1.00000/(1.00000+exp((STATES[V] - 213.600)/151.200));
ALGEBRAIC[AiS] = 1.00000 - ALGEBRAIC[AiF];
ALGEBRAIC[i] =  ALGEBRAIC[AiF]*STATES[iF]+ ALGEBRAIC[AiS]*STATES[iS];
ALGEBRAIC[assp] = 1.00000/(1.00000+exp(- (STATES[V] - 24.3400)/14.8200));
ALGEBRAIC[dti_develop] = 1.35400+0.000100000/(exp((STATES[V] - 167.400)/15.8900)+exp(- (STATES[V] - 12.2300)/0.215400));
ALGEBRAIC[dti_recover] = 1.00000 - 0.500000/(1.00000+exp((STATES[V]+70.0000)/20.0000));
ALGEBRAIC[tiFp] = ALGEBRAIC[dti_develop] * ALGEBRAIC[dti_recover] * ALGEBRAIC[tiF];
ALGEBRAIC[tiSp] = ALGEBRAIC[dti_develop] * ALGEBRAIC[dti_recover] * ALGEBRAIC[tiS];
ALGEBRAIC[ip] =  ALGEBRAIC[AiF]*STATES[iFp]+ ALGEBRAIC[AiS]*STATES[iSp];
ALGEBRAIC[EK] =  (( CONSTANTS[R]*CONSTANTS[T])/CONSTANTS[F])*log(CONSTANTS[ko]/STATES[ki]);
ALGEBRAIC[fItop] = 1.00000/(1.00000+CONSTANTS[KmCaMK]/ALGEBRAIC[CaMKa]);
ALGEBRAIC[Ito] =  CONSTANTS[Gto]*(STATES[V] - ALGEBRAIC[EK])*( (1.00000 - ALGEBRAIC[fItop])*STATES[a]*ALGEBRAIC[i]+ ALGEBRAIC[fItop]*STATES[ap]*ALGEBRAIC[ip]);
ALGEBRAIC[xrss] = 1.00000/(1.00000+exp(- (STATES[V]+8.33700)/6.78900));
ALGEBRAIC[txrf] = 12.9800+1.00000/( 0.365200*exp((STATES[V] - 31.6600)/3.86900)+ 4.12300e-05*exp(- (STATES[V] - 47.7800)/20.3800));
ALGEBRAIC[txrs] = 1.86500+1.00000/( 0.0662900*exp((STATES[V] - 34.7000)/7.35500)+ 1.12800e-05*exp(- (STATES[V] - 29.7400)/25.9400));
ALGEBRAIC[Axrf] = 1.00000/(1.00000+exp((STATES[V]+54.8100)/38.2100));
ALGEBRAIC[Axrs] = 1.00000 - ALGEBRAIC[Axrf];
ALGEBRAIC[xr] =  ALGEBRAIC[Axrf]*STATES[xrf]+ ALGEBRAIC[Axrs]*STATES[xrs];
ALGEBRAIC[rkr] = ( (1.00000/(1.00000+exp((STATES[V]+55.0000)/75.0000)))*1.00000)/(1.00000+exp((STATES[V] - 10.0000)/30.0000));
ALGEBRAIC[IKr] =  CONSTANTS[GKr]* pow((CONSTANTS[ko]/5.40000), 1.0 / 2)*ALGEBRAIC[xr]*ALGEBRAIC[rkr]*(STATES[V] - ALGEBRAIC[EK]);
ALGEBRAIC[xs1ss] = 1.00000/(1.00000+exp(- (STATES[V]+11.6000)/8.93200));
ALGEBRAIC[txs1] = 817.300+1.00000/( 0.000232600*exp((STATES[V]+48.2800)/17.8000)+ 0.00129200*exp(- (STATES[V]+210.000)/230.000));
ALGEBRAIC[xs2ss] = ALGEBRAIC[xs1ss];
ALGEBRAIC[txs2] = 1.00000/( 0.0100000*exp((STATES[V] - 50.0000)/20.0000)+ 0.0193000*exp(- (STATES[V]+66.5400)/31.0000));
ALGEBRAIC[KsCa] = 1.00000+0.600000/(1.00000+pow(3.80000e-05/STATES[cai], 1.40000));
ALGEBRAIC[EKs] =  (( CONSTANTS[R]*CONSTANTS[T])/CONSTANTS[F])*log((CONSTANTS[ko]+ CONSTANTS[PKNa]*CONSTANTS[nao])/(STATES[ki]+ CONSTANTS[PKNa]*STATES[nai]));
ALGEBRAIC[IKs] =  CONSTANTS[GKs]*ALGEBRAIC[KsCa]*STATES[xs1]*STATES[xs2]*(STATES[V] - ALGEBRAIC[EKs]);
ALGEBRAIC[xk1ss] = 1.00000/(1.00000+exp(- (STATES[V]+ 2.55380*CONSTANTS[ko]+144.590)/( 1.56920*CONSTANTS[ko]+3.81150)));
ALGEBRAIC[txk1] = 122.200/(exp(- (STATES[V]+127.200)/20.3600)+exp((STATES[V]+236.800)/69.3300));
ALGEBRAIC[rk1] = 1.00000/(1.00000+exp(((STATES[V]+105.800) -  2.60000*CONSTANTS[ko])/9.49300));
ALGEBRAIC[IK1] =  CONSTANTS[GK1]* pow(CONSTANTS[ko], 1.0 / 2)*ALGEBRAIC[rk1]*STATES[xk1]*(STATES[V] - ALGEBRAIC[EK]);
ALGEBRAIC[xkb] = 1.00000/(1.00000+exp(- (STATES[V] - 14.4800)/18.3400));
ALGEBRAIC[IKb] =  CONSTANTS[GKb]*ALGEBRAIC[xkb]*(STATES[V] - ALGEBRAIC[EK]);
ALGEBRAIC[dss] = 1.00000/(1.00000+exp(- (STATES[V]+3.94000)/4.23000));
ALGEBRAIC[td] = 0.600000+1.00000/(exp( - 0.0500000*(STATES[V]+6.00000))+exp( 0.0900000*(STATES[V]+14.0000)));
ALGEBRAIC[fss] = 1.00000/(1.00000+exp((STATES[V]+19.5800)/3.69600));
ALGEBRAIC[tff] = 7.00000+1.00000/( 0.00450000*exp(- (STATES[V]+20.0000)/10.0000)+ 0.00450000*exp((STATES[V]+20.0000)/10.0000));
ALGEBRAIC[tfs] = 1000.00+1.00000/( 3.50000e-05*exp(- (STATES[V]+5.00000)/4.00000)+ 3.50000e-05*exp((STATES[V]+5.00000)/6.00000));
ALGEBRAIC[f] =  CONSTANTS[Aff]*STATES[ff]+ CONSTANTS[Afs]*STATES[fs];
ALGEBRAIC[fcass] = ALGEBRAIC[fss];
ALGEBRAIC[tfcaf] = 7.00000+1.00000/( 0.0400000*exp(- (STATES[V] - 4.00000)/7.00000)+ 0.0400000*exp((STATES[V] - 4.00000)/7.00000));
ALGEBRAIC[tfcas] = 100.000+1.00000/( 0.000120000*exp(- STATES[V]/3.00000)+ 0.000120000*exp(STATES[V]/7.00000));
ALGEBRAIC[Afcaf] = 0.300000+0.600000/(1.00000+exp((STATES[V] - 10.0000)/10.0000));
ALGEBRAIC[Afcas] = 1.00000 - ALGEBRAIC[Afcaf];
ALGEBRAIC[fca] =  ALGEBRAIC[Afcaf]*STATES[fcaf]+ ALGEBRAIC[Afcas]*STATES[fcas];
ALGEBRAIC[tffp] =  2.50000*ALGEBRAIC[tff];
ALGEBRAIC[fp] =  CONSTANTS[Aff]*STATES[ffp]+ CONSTANTS[Afs]*STATES[fs];
ALGEBRAIC[tfcafp] =  2.50000*ALGEBRAIC[tfcaf];
ALGEBRAIC[fcap] =  ALGEBRAIC[Afcaf]*STATES[fcafp]+ ALGEBRAIC[Afcas]*STATES[fcas];
ALGEBRAIC[km2n] =  STATES[jca]*1.00000;
ALGEBRAIC[anca] = 1.00000/(CONSTANTS[k2n]/ALGEBRAIC[km2n]+pow(1.00000+CONSTANTS[Kmn]/STATES[cass], 4.00000));
ALGEBRAIC[PhiCaL] = ( 4.00000*ALGEBRAIC[vffrt]*( STATES[cass]*exp( 2.00000*ALGEBRAIC[vfrt]) -  0.341000*CONSTANTS[cao]))/(exp( 2.00000*ALGEBRAIC[vfrt]) - 1.00000);
ALGEBRAIC[PhiCaNa] = ( 1.00000*ALGEBRAIC[vffrt]*( 0.750000*STATES[nass]*exp( 1.00000*ALGEBRAIC[vfrt]) -  0.750000*CONSTANTS[nao]))/(exp( 1.00000*ALGEBRAIC[vfrt]) - 1.00000);
ALGEBRAIC[PhiCaK] = ( 1.00000*ALGEBRAIC[vffrt]*( 0.750000*STATES[kss]*exp( 1.00000*ALGEBRAIC[vfrt]) -  0.750000*CONSTANTS[ko]))/(exp( 1.00000*ALGEBRAIC[vfrt]) - 1.00000);
ALGEBRAIC[fICaLp] = 1.00000/(1.00000+CONSTANTS[KmCaMK]/ALGEBRAIC[CaMKa]);
ALGEBRAIC[ICaL] =  (1.00000 - ALGEBRAIC[fICaLp])*CONSTANTS[PCa]*ALGEBRAIC[PhiCaL]*STATES[d]*( ALGEBRAIC[f]*(1.00000 - STATES[nca])+ STATES[jca]*ALGEBRAIC[fca]*STATES[nca])+ ALGEBRAIC[fICaLp]*CONSTANTS[PCap]*ALGEBRAIC[PhiCaL]*STATES[d]*( ALGEBRAIC[fp]*(1.00000 - STATES[nca])+ STATES[jca]*ALGEBRAIC[fcap]*STATES[nca]);
ALGEBRAIC[ICaNa] =  (1.00000 - ALGEBRAIC[fICaLp])*CONSTANTS[PCaNa]*ALGEBRAIC[PhiCaNa]*STATES[d]*( ALGEBRAIC[f]*(1.00000 - STATES[nca])+ STATES[jca]*ALGEBRAIC[fca]*STATES[nca])+ ALGEBRAIC[fICaLp]*CONSTANTS[PCaNap]*ALGEBRAIC[PhiCaNa]*STATES[d]*( ALGEBRAIC[fp]*(1.00000 - STATES[nca])+ STATES[jca]*ALGEBRAIC[fcap]*STATES[nca]);
ALGEBRAIC[ICaK] =  (1.00000 - ALGEBRAIC[fICaLp])*CONSTANTS[PCaK]*ALGEBRAIC[PhiCaK]*STATES[d]*( ALGEBRAIC[f]*(1.00000 - STATES[nca])+ STATES[jca]*ALGEBRAIC[fca]*STATES[nca])+ ALGEBRAIC[fICaLp]*CONSTANTS[PCaKp]*ALGEBRAIC[PhiCaK]*STATES[d]*( ALGEBRAIC[fp]*(1.00000 - STATES[nca])+ STATES[jca]*ALGEBRAIC[fcap]*STATES[nca]);
ALGEBRAIC[ICab] = ( CONSTANTS[PCab]*4.00000*ALGEBRAIC[vffrt]*( STATES[cai]*exp( 2.00000*ALGEBRAIC[vfrt]) -  0.341000*CONSTANTS[cao]))/(exp( 2.00000*ALGEBRAIC[vfrt]) - 1.00000);
ALGEBRAIC[IpCa] = ( CONSTANTS[GpCa]*STATES[cai])/(CONSTANTS[KmCap]+STATES[cai]);
ALGEBRAIC[hna] = exp(( CONSTANTS[qna]*STATES[V]*CONSTANTS[F])/( CONSTANTS[R]*CONSTANTS[T]));
ALGEBRAIC[hca] = exp(( CONSTANTS[qca]*STATES[V]*CONSTANTS[F])/( CONSTANTS[R]*CONSTANTS[T]));
ALGEBRAIC[h1_i] = 1.00000+ (STATES[nai]/CONSTANTS[kna3])*(1.00000+ALGEBRAIC[hna]);
ALGEBRAIC[h2_i] = ( STATES[nai]*ALGEBRAIC[hna])/( CONSTANTS[kna3]*ALGEBRAIC[h1_i]);
ALGEBRAIC[h3_i] = 1.00000/ALGEBRAIC[h1_i];
ALGEBRAIC[h4_i] = 1.00000+ (STATES[nai]/CONSTANTS[kna1])*(1.00000+STATES[nai]/CONSTANTS[kna2]);
ALGEBRAIC[h5_i] = ( STATES[nai]*STATES[nai])/( ALGEBRAIC[h4_i]*CONSTANTS[kna1]*CONSTANTS[kna2]);
ALGEBRAIC[h6_i] = 1.00000/ALGEBRAIC[h4_i];
ALGEBRAIC[h7_i] = 1.00000+ (CONSTANTS[nao]/CONSTANTS[kna3])*(1.00000+1.00000/ALGEBRAIC[hna]);
ALGEBRAIC[h8_i] = CONSTANTS[nao]/( CONSTANTS[kna3]*ALGEBRAIC[hna]*ALGEBRAIC[h7_i]);
ALGEBRAIC[h9_i] = 1.00000/ALGEBRAIC[h7_i];
ALGEBRAIC[k3p_i] =  ALGEBRAIC[h9_i]*CONSTANTS[wca];
ALGEBRAIC[k3pp_i] =  ALGEBRAIC[h8_i]*CONSTANTS[wnaca];
ALGEBRAIC[k3_i] = ALGEBRAIC[k3p_i]+ALGEBRAIC[k3pp_i];
ALGEBRAIC[k4p_i] = ( ALGEBRAIC[h3_i]*CONSTANTS[wca])/ALGEBRAIC[hca];
ALGEBRAIC[k4pp_i] =  ALGEBRAIC[h2_i]*CONSTANTS[wnaca];
ALGEBRAIC[k4_i] = ALGEBRAIC[k4p_i]+ALGEBRAIC[k4pp_i];
ALGEBRAIC[k6_i] =  ALGEBRAIC[h6_i]*STATES[cai]*CONSTANTS[kcaon];
ALGEBRAIC[k7_i] =  ALGEBRAIC[h5_i]*ALGEBRAIC[h2_i]*CONSTANTS[wna];
ALGEBRAIC[k8_i] =  ALGEBRAIC[h8_i]*CONSTANTS[h11_i]*CONSTANTS[wna];
ALGEBRAIC[x1_i] =  CONSTANTS[k2_i]*ALGEBRAIC[k4_i]*(ALGEBRAIC[k7_i]+ALGEBRAIC[k6_i])+ CONSTANTS[k5_i]*ALGEBRAIC[k7_i]*(CONSTANTS[k2_i]+ALGEBRAIC[k3_i]);
ALGEBRAIC[x2_i] =  CONSTANTS[k1_i]*ALGEBRAIC[k7_i]*(ALGEBRAIC[k4_i]+CONSTANTS[k5_i])+ ALGEBRAIC[k4_i]*ALGEBRAIC[k6_i]*(CONSTANTS[k1_i]+ALGEBRAIC[k8_i]);
ALGEBRAIC[x3_i] =  CONSTANTS[k1_i]*ALGEBRAIC[k3_i]*(ALGEBRAIC[k7_i]+ALGEBRAIC[k6_i])+ ALGEBRAIC[k8_i]*ALGEBRAIC[k6_i]*(CONSTANTS[k2_i]+ALGEBRAIC[k3_i]);
ALGEBRAIC[x4_i] =  CONSTANTS[k2_i]*ALGEBRAIC[k8_i]*(ALGEBRAIC[k4_i]+CONSTANTS[k5_i])+ ALGEBRAIC[k3_i]*CONSTANTS[k5_i]*(CONSTANTS[k1_i]+ALGEBRAIC[k8_i]);
ALGEBRAIC[E1_i] = ALGEBRAIC[x1_i]/(ALGEBRAIC[x1_i]+ALGEBRAIC[x2_i]+ALGEBRAIC[x3_i]+ALGEBRAIC[x4_i]);
ALGEBRAIC[E2_i] = ALGEBRAIC[x2_i]/(ALGEBRAIC[x1_i]+ALGEBRAIC[x2_i]+ALGEBRAIC[x3_i]+ALGEBRAIC[x4_i]);
ALGEBRAIC[E3_i] = ALGEBRAIC[x3_i]/(ALGEBRAIC[x1_i]+ALGEBRAIC[x2_i]+ALGEBRAIC[x3_i]+ALGEBRAIC[x4_i]);
ALGEBRAIC[E4_i] = ALGEBRAIC[x4_i]/(ALGEBRAIC[x1_i]+ALGEBRAIC[x2_i]+ALGEBRAIC[x3_i]+ALGEBRAIC[x4_i]);
ALGEBRAIC[allo_i] = 1.00000/(1.00000+pow(CONSTANTS[KmCaAct]/STATES[cai], 2.00000));
ALGEBRAIC[JncxCa_i] =  ALGEBRAIC[E2_i]*CONSTANTS[k2_i] -  ALGEBRAIC[E1_i]*CONSTANTS[k1_i];
ALGEBRAIC[JncxNa_i] = ( 3.00000*( ALGEBRAIC[E4_i]*ALGEBRAIC[k7_i] -  ALGEBRAIC[E1_i]*ALGEBRAIC[k8_i])+ ALGEBRAIC[E3_i]*ALGEBRAIC[k4pp_i]) -  ALGEBRAIC[E2_i]*ALGEBRAIC[k3pp_i];
ALGEBRAIC[INaCa_i] =  0.800000*CONSTANTS[Gncx]*ALGEBRAIC[allo_i]*( CONSTANTS[zna]*ALGEBRAIC[JncxNa_i]+ CONSTANTS[zca]*ALGEBRAIC[JncxCa_i]);
ALGEBRAIC[h1_ss] = 1.00000+ (STATES[nass]/CONSTANTS[kna3])*(1.00000+ALGEBRAIC[hna]);
ALGEBRAIC[h2_ss] = ( STATES[nass]*ALGEBRAIC[hna])/( CONSTANTS[kna3]*ALGEBRAIC[h1_ss]);
ALGEBRAIC[h3_ss] = 1.00000/ALGEBRAIC[h1_ss];
ALGEBRAIC[h4_ss] = 1.00000+ (STATES[nass]/CONSTANTS[kna1])*(1.00000+STATES[nass]/CONSTANTS[kna2]);
ALGEBRAIC[h5_ss] = ( STATES[nass]*STATES[nass])/( ALGEBRAIC[h4_ss]*CONSTANTS[kna1]*CONSTANTS[kna2]);
ALGEBRAIC[h6_ss] = 1.00000/ALGEBRAIC[h4_ss];
ALGEBRAIC[h7_ss] = 1.00000+ (CONSTANTS[nao]/CONSTANTS[kna3])*(1.00000+1.00000/ALGEBRAIC[hna]);
ALGEBRAIC[h8_ss] = CONSTANTS[nao]/( CONSTANTS[kna3]*ALGEBRAIC[hna]*ALGEBRAIC[h7_ss]);
ALGEBRAIC[h9_ss] = 1.00000/ALGEBRAIC[h7_ss];
ALGEBRAIC[k3p_ss] =  ALGEBRAIC[h9_ss]*CONSTANTS[wca];
ALGEBRAIC[k3pp_ss] =  ALGEBRAIC[h8_ss]*CONSTANTS[wnaca];
ALGEBRAIC[k3_ss] = ALGEBRAIC[k3p_ss]+ALGEBRAIC[k3pp_ss];
ALGEBRAIC[k4p_ss] = ( ALGEBRAIC[h3_ss]*CONSTANTS[wca])/ALGEBRAIC[hca];
ALGEBRAIC[k4pp_ss] =  ALGEBRAIC[h2_ss]*CONSTANTS[wnaca];
ALGEBRAIC[k4_ss] = ALGEBRAIC[k4p_ss]+ALGEBRAIC[k4pp_ss];
ALGEBRAIC[k6_ss] =  ALGEBRAIC[h6_ss]*STATES[cass]*CONSTANTS[kcaon];
ALGEBRAIC[k7_ss] =  ALGEBRAIC[h5_ss]*ALGEBRAIC[h2_ss]*CONSTANTS[wna];
ALGEBRAIC[k8_ss] =  ALGEBRAIC[h8_ss]*CONSTANTS[h11_ss]*CONSTANTS[wna];
ALGEBRAIC[x1_ss] =  CONSTANTS[k2_ss]*ALGEBRAIC[k4_ss]*(ALGEBRAIC[k7_ss]+ALGEBRAIC[k6_ss])+ CONSTANTS[k5_ss]*ALGEBRAIC[k7_ss]*(CONSTANTS[k2_ss]+ALGEBRAIC[k3_ss]);
ALGEBRAIC[x2_ss] =  CONSTANTS[k1_ss]*ALGEBRAIC[k7_ss]*(ALGEBRAIC[k4_ss]+CONSTANTS[k5_ss])+ ALGEBRAIC[k4_ss]*ALGEBRAIC[k6_ss]*(CONSTANTS[k1_ss]+ALGEBRAIC[k8_ss]);
ALGEBRAIC[x3_ss] =  CONSTANTS[k1_ss]*ALGEBRAIC[k3_ss]*(ALGEBRAIC[k7_ss]+ALGEBRAIC[k6_ss])+ ALGEBRAIC[k8_ss]*ALGEBRAIC[k6_ss]*(CONSTANTS[k2_ss]+ALGEBRAIC[k3_ss]);
ALGEBRAIC[x4_ss] =  CONSTANTS[k2_ss]*ALGEBRAIC[k8_ss]*(ALGEBRAIC[k4_ss]+CONSTANTS[k5_ss])+ ALGEBRAIC[k3_ss]*CONSTANTS[k5_ss]*(CONSTANTS[k1_ss]+ALGEBRAIC[k8_ss]);
ALGEBRAIC[E1_ss] = ALGEBRAIC[x1_ss]/(ALGEBRAIC[x1_ss]+ALGEBRAIC[x2_ss]+ALGEBRAIC[x3_ss]+ALGEBRAIC[x4_ss]);
ALGEBRAIC[E2_ss] = ALGEBRAIC[x2_ss]/(ALGEBRAIC[x1_ss]+ALGEBRAIC[x2_ss]+ALGEBRAIC[x3_ss]+ALGEBRAIC[x4_ss]);
ALGEBRAIC[E3_ss] = ALGEBRAIC[x3_ss]/(ALGEBRAIC[x1_ss]+ALGEBRAIC[x2_ss]+ALGEBRAIC[x3_ss]+ALGEBRAIC[x4_ss]);
ALGEBRAIC[E4_ss] = ALGEBRAIC[x4_ss]/(ALGEBRAIC[x1_ss]+ALGEBRAIC[x2_ss]+ALGEBRAIC[x3_ss]+ALGEBRAIC[x4_ss]);
ALGEBRAIC[allo_ss] = 1.00000/(1.00000+pow(CONSTANTS[KmCaAct]/STATES[cass], 2.00000));
ALGEBRAIC[JncxCa_ss] =  ALGEBRAIC[E2_ss]*CONSTANTS[k2_ss] -  ALGEBRAIC[E1_ss]*CONSTANTS[k1_ss];
ALGEBRAIC[JncxNa_ss] = ( 3.00000*( ALGEBRAIC[E4_ss]*ALGEBRAIC[k7_ss] -  ALGEBRAIC[E1_ss]*ALGEBRAIC[k8_ss])+ ALGEBRAIC[E3_ss]*ALGEBRAIC[k4pp_ss]) -  ALGEBRAIC[E2_ss]*ALGEBRAIC[k3pp_ss];
ALGEBRAIC[INaCa_ss] =  0.200000*CONSTANTS[Gncx]*ALGEBRAIC[allo_ss]*( CONSTANTS[zna]*ALGEBRAIC[JncxNa_ss]+ CONSTANTS[zca]*ALGEBRAIC[JncxCa_ss]);
ALGEBRAIC[Knai] =  CONSTANTS[Knai0]*exp(( CONSTANTS[delta]*STATES[V]*CONSTANTS[F])/( 3.00000*CONSTANTS[R]*CONSTANTS[T]));
ALGEBRAIC[Knao] =  CONSTANTS[Knao0]*exp(( (1.00000 - CONSTANTS[delta])*STATES[V]*CONSTANTS[F])/( 3.00000*CONSTANTS[R]*CONSTANTS[T]));
ALGEBRAIC[P] = CONSTANTS[eP]/(1.00000+CONSTANTS[H]/CONSTANTS[Khp]+STATES[nai]/CONSTANTS[Knap]+STATES[ki]/CONSTANTS[Kxkur]);
ALGEBRAIC[a1] = ( CONSTANTS[k1p]*pow(STATES[nai]/ALGEBRAIC[Knai], 3.00000))/((pow(1.00000+STATES[nai]/ALGEBRAIC[Knai], 3.00000)+pow(1.00000+STATES[ki]/CONSTANTS[Kki], 2.00000)) - 1.00000);
ALGEBRAIC[a3] = ( CONSTANTS[k3p]*pow(CONSTANTS[ko]/CONSTANTS[Kko], 2.00000))/((pow(1.00000+CONSTANTS[nao]/ALGEBRAIC[Knao], 3.00000)+pow(1.00000+CONSTANTS[ko]/CONSTANTS[Kko], 2.00000)) - 1.00000);
ALGEBRAIC[b2] = ( CONSTANTS[k2m]*pow(CONSTANTS[nao]/ALGEBRAIC[Knao], 3.00000))/((pow(1.00000+CONSTANTS[nao]/ALGEBRAIC[Knao], 3.00000)+pow(1.00000+CONSTANTS[ko]/CONSTANTS[Kko], 2.00000)) - 1.00000);
ALGEBRAIC[b3] = ( CONSTANTS[k3m]*ALGEBRAIC[P]*CONSTANTS[H])/(1.00000+CONSTANTS[MgATP]/CONSTANTS[Kmgatp]);
ALGEBRAIC[b4] = ( CONSTANTS[k4m]*pow(STATES[ki]/CONSTANTS[Kki], 2.00000))/((pow(1.00000+STATES[nai]/ALGEBRAIC[Knai], 3.00000)+pow(1.00000+STATES[ki]/CONSTANTS[Kki], 2.00000)) - 1.00000);
ALGEBRAIC[x1] =  CONSTANTS[a4]*ALGEBRAIC[a1]*CONSTANTS[a2]+ ALGEBRAIC[b2]*ALGEBRAIC[b4]*ALGEBRAIC[b3]+ CONSTANTS[a2]*ALGEBRAIC[b4]*ALGEBRAIC[b3]+ ALGEBRAIC[b3]*ALGEBRAIC[a1]*CONSTANTS[a2];
ALGEBRAIC[x2] =  ALGEBRAIC[b2]*CONSTANTS[b1]*ALGEBRAIC[b4]+ ALGEBRAIC[a1]*CONSTANTS[a2]*ALGEBRAIC[a3]+ ALGEBRAIC[a3]*CONSTANTS[b1]*ALGEBRAIC[b4]+ CONSTANTS[a2]*ALGEBRAIC[a3]*ALGEBRAIC[b4];
ALGEBRAIC[x3] =  CONSTANTS[a2]*ALGEBRAIC[a3]*CONSTANTS[a4]+ ALGEBRAIC[b3]*ALGEBRAIC[b2]*CONSTANTS[b1]+ ALGEBRAIC[b2]*CONSTANTS[b1]*CONSTANTS[a4]+ ALGEBRAIC[a3]*CONSTANTS[a4]*CONSTANTS[b1];
ALGEBRAIC[x4] =  ALGEBRAIC[b4]*ALGEBRAIC[b3]*ALGEBRAIC[b2]+ ALGEBRAIC[a3]*CONSTANTS[a4]*ALGEBRAIC[a1]+ ALGEBRAIC[b2]*CONSTANTS[a4]*ALGEBRAIC[a1]+ ALGEBRAIC[b3]*ALGEBRAIC[b2]*ALGEBRAIC[a1];
ALGEBRAIC[E1] = ALGEBRAIC[x1]/(ALGEBRAIC[x1]+ALGEBRAIC[x2]+ALGEBRAIC[x3]+ALGEBRAIC[x4]);
ALGEBRAIC[E2] = ALGEBRAIC[x2]/(ALGEBRAIC[x1]+ALGEBRAIC[x2]+ALGEBRAIC[x3]+ALGEBRAIC[x4]);
ALGEBRAIC[E4] = ALGEBRAIC[x4]/(ALGEBRAIC[x1]+ALGEBRAIC[x2]+ALGEBRAIC[x3]+ALGEBRAIC[x4]);
ALGEBRAIC[E3] = ALGEBRAIC[x3]/(ALGEBRAIC[x1]+ALGEBRAIC[x2]+ALGEBRAIC[x3]+ALGEBRAIC[x4]);
ALGEBRAIC[JnakNa] =  3.00000*( ALGEBRAIC[E1]*ALGEBRAIC[a3] -  ALGEBRAIC[E2]*ALGEBRAIC[b3]);
ALGEBRAIC[JnakK] =  2.00000*( ALGEBRAIC[E4]*CONSTANTS[b1] -  ALGEBRAIC[E3]*ALGEBRAIC[a1]);
ALGEBRAIC[INaK] =  CONSTANTS[Pnak]*( CONSTANTS[zna]*ALGEBRAIC[JnakNa]+ CONSTANTS[zk]*ALGEBRAIC[JnakK]);
ALGEBRAIC[Jdiff] = (STATES[cass] - STATES[cai])/0.200000;
ALGEBRAIC[JdiffK] = (STATES[kss] - STATES[ki])/2.00000;
ALGEBRAIC[JdiffNa] = (STATES[nass] - STATES[nai])/2.00000;
ALGEBRAIC[Jupnp] = ( CONSTANTS[upScale]*0.00437500*STATES[cai])/(STATES[cai]+0.000920000);
ALGEBRAIC[Jupp] = ( CONSTANTS[upScale]*2.75000*0.00437500*STATES[cai])/((STATES[cai]+0.000920000) - 0.000170000);
ALGEBRAIC[fJupp] = 1.00000/(1.00000+CONSTANTS[KmCaMK]/ALGEBRAIC[CaMKa]);
ALGEBRAIC[Jleak] = ( 0.00393750*STATES[cansr])/15.0000;
ALGEBRAIC[Jup] = ( (1.00000 - ALGEBRAIC[fJupp])*ALGEBRAIC[Jupnp]+ ALGEBRAIC[fJupp]*ALGEBRAIC[Jupp]) - ALGEBRAIC[Jleak];
ALGEBRAIC[fJrelp] = 1.00000/(1.00000+CONSTANTS[KmCaMK]/ALGEBRAIC[CaMKa]);
ALGEBRAIC[Jrel_inf_temp] = ( CONSTANTS[a_rel]*- ALGEBRAIC[ICaL])/(1.00000+ 1.00000*pow(1.50000/STATES[cajsr], 8.00000));
ALGEBRAIC[Jrel_inf] = (CONSTANTS[celltype]==2.00000 ?  ALGEBRAIC[Jrel_inf_temp]*1.70000 : ALGEBRAIC[Jrel_inf_temp]);
ALGEBRAIC[tau_relp_temp] = CONSTANTS[bt]/(1.00000+0.0123000/STATES[cajsr]);
ALGEBRAIC[tau_rel] = (ALGEBRAIC[tau_rel_temp]<0.00100000 ? 0.00100000 : ALGEBRAIC[tau_rel_temp]);
ALGEBRAIC[Jrel_temp] = ( CONSTANTS[a_relp]*- ALGEBRAIC[ICaL])/(1.00000+pow(1.50000/STATES[cajsr], 8.00000));
ALGEBRAIC[Jrel_infp] = (CONSTANTS[celltype]==2.00000 ?  ALGEBRAIC[Jrel_temp]*1.70000 : ALGEBRAIC[Jrel_temp]);
ALGEBRAIC[tau_rel_temp] = CONSTANTS[bt]/(1.00000+0.0123000/STATES[cajsr]);
ALGEBRAIC[tau_relp] = (ALGEBRAIC[tau_relp_temp]<0.00100000 ? 0.00100000 : ALGEBRAIC[tau_relp_temp]);
ALGEBRAIC[Jrel] =  (1.00000 - ALGEBRAIC[fJrelp])*STATES[Jrelnp]+ ALGEBRAIC[fJrelp]*STATES[Jrelp];
ALGEBRAIC[Jtr] = (STATES[cansr] - STATES[cajsr])/100.000;
ALGEBRAIC[Bcai] = 1.00000/(1.00000+( CONSTANTS[cmdnmax]*CONSTANTS[kmcmdn])/pow(CONSTANTS[kmcmdn]+STATES[cai], 2.00000)+( CONSTANTS[trpnmax]*CONSTANTS[kmtrpn])/pow(CONSTANTS[kmtrpn]+STATES[cai], 2.00000));
ALGEBRAIC[Bcass] = 1.00000/(1.00000+( CONSTANTS[BSRmax]*CONSTANTS[KmBSR])/pow(CONSTANTS[KmBSR]+STATES[cass], 2.00000)+( CONSTANTS[BSLmax]*CONSTANTS[KmBSL])/pow(CONSTANTS[KmBSL]+STATES[cass], 2.00000));
ALGEBRAIC[Bcajsr] = 1.00000/(1.00000+( CONSTANTS[csqnmax]*CONSTANTS[kmcsqn])/pow(CONSTANTS[kmcsqn]+STATES[cajsr], 2.00000));
RATES[m] = (ALGEBRAIC[mss] - STATES[m])/ALGEBRAIC[tm];
RATES[j] = (ALGEBRAIC[jss] - STATES[j])/ALGEBRAIC[tj];
RATES[jp] = (ALGEBRAIC[jss] - STATES[jp])/ALGEBRAIC[tjp];
RATES[hf] = (ALGEBRAIC[hss] - STATES[hf])/ALGEBRAIC[thf];
RATES[hs] = (ALGEBRAIC[hss] - STATES[hs])/ALGEBRAIC[ths];
RATES[hsp] = (ALGEBRAIC[hssp] - STATES[hsp])/ALGEBRAIC[thsp];
RATES[mL] = (ALGEBRAIC[mLss] - STATES[mL])/ALGEBRAIC[tmL];
RATES[hL] = (ALGEBRAIC[hLss] - STATES[hL])/CONSTANTS[thL];
RATES[hLp] = (ALGEBRAIC[hLssp] - STATES[hLp])/CONSTANTS[thLp];
RATES[a] = (ALGEBRAIC[ass] - STATES[a])/ALGEBRAIC[ta];
RATES[ap] = (ALGEBRAIC[assp] - STATES[ap])/ALGEBRAIC[ta];
RATES[iF] = (ALGEBRAIC[iss] - STATES[iF])/ALGEBRAIC[tiF];
RATES[iS] = (ALGEBRAIC[iss] - STATES[iS])/ALGEBRAIC[tiS];
RATES[iFp] = (ALGEBRAIC[iss] - STATES[iFp])/ALGEBRAIC[tiFp];
RATES[iSp] = (ALGEBRAIC[iss] - STATES[iSp])/ALGEBRAIC[tiSp];
RATES[xrf] = (ALGEBRAIC[xrss] - STATES[xrf])/ALGEBRAIC[txrf];
RATES[xrs] = (ALGEBRAIC[xrss] - STATES[xrs])/ALGEBRAIC[txrs];
RATES[xs1] = (ALGEBRAIC[xs1ss] - STATES[xs1])/ALGEBRAIC[txs1];
RATES[xs2] = (ALGEBRAIC[xs2ss] - STATES[xs2])/ALGEBRAIC[txs2];
RATES[xk1] = (ALGEBRAIC[xk1ss] - STATES[xk1])/ALGEBRAIC[txk1];
RATES[d] = (ALGEBRAIC[dss] - STATES[d])/ALGEBRAIC[td];
RATES[ff] = (ALGEBRAIC[fss] - STATES[ff])/ALGEBRAIC[tff];
RATES[fs] = (ALGEBRAIC[fss] - STATES[fs])/ALGEBRAIC[tfs];
RATES[fcaf] = (ALGEBRAIC[fcass] - STATES[fcaf])/ALGEBRAIC[tfcaf];
RATES[nca] =  ALGEBRAIC[anca]*CONSTANTS[k2n] -  STATES[nca]*ALGEBRAIC[km2n];
RATES[jca] = (ALGEBRAIC[fcass] - STATES[jca])/CONSTANTS[tjca];
RATES[fcas] = (ALGEBRAIC[fcass] - STATES[fcas])/ALGEBRAIC[tfcas];
RATES[ffp] = (ALGEBRAIC[fss] - STATES[ffp])/ALGEBRAIC[tffp];
RATES[fcafp] = (ALGEBRAIC[fcass] - STATES[fcafp])/ALGEBRAIC[tfcafp];
RATES[Jrelnp] = (ALGEBRAIC[Jrel_inf] - STATES[Jrelnp])/ALGEBRAIC[tau_rel];
RATES[Jrelp] = (ALGEBRAIC[Jrel_infp] - STATES[Jrelp])/ALGEBRAIC[tau_relp];
RATES[CaMKt] =  CONSTANTS[aCaMK]*ALGEBRAIC[CaMKb]*(ALGEBRAIC[CaMKb]+STATES[CaMKt]) -  CONSTANTS[bCaMK]*STATES[CaMKt];
RATES[nai] = ( - (ALGEBRAIC[INa]+ALGEBRAIC[INaL]+ 3.00000*ALGEBRAIC[INaCa_i]+ 3.00000*ALGEBRAIC[INaK]+ALGEBRAIC[INab])*CONSTANTS[Acap]*CONSTANTS[cm])/( CONSTANTS[F]*CONSTANTS[vmyo])+( ALGEBRAIC[JdiffNa]*CONSTANTS[vss])/CONSTANTS[vmyo];
RATES[nass] = ( - (ALGEBRAIC[ICaNa]+ 3.00000*ALGEBRAIC[INaCa_ss])*CONSTANTS[cm]*CONSTANTS[Acap])/( CONSTANTS[F]*CONSTANTS[vss]) - ALGEBRAIC[JdiffNa];
RATES[ki] = ( - ((ALGEBRAIC[Ito]+ALGEBRAIC[IKr]+ALGEBRAIC[IKs]+ALGEBRAIC[IK1]+ALGEBRAIC[IKb]+ALGEBRAIC[Istim]) -  2.00000*ALGEBRAIC[INaK])*CONSTANTS[cm]*CONSTANTS[Acap])/( CONSTANTS[F]*CONSTANTS[vmyo])+( ALGEBRAIC[JdiffK]*CONSTANTS[vss])/CONSTANTS[vmyo];
RATES[kss] = ( - ALGEBRAIC[ICaK]*CONSTANTS[cm]*CONSTANTS[Acap])/( CONSTANTS[F]*CONSTANTS[vss]) - ALGEBRAIC[JdiffK];
RATES[cai] =  ALGEBRAIC[Bcai]*((( - ((ALGEBRAIC[IpCa]+ALGEBRAIC[ICab]) -  2.00000*ALGEBRAIC[INaCa_i])*CONSTANTS[cm]*CONSTANTS[Acap])/( 2.00000*CONSTANTS[F]*CONSTANTS[vmyo]) - ( ALGEBRAIC[Jup]*CONSTANTS[vnsr])/CONSTANTS[vmyo])+( ALGEBRAIC[Jdiff]*CONSTANTS[vss])/CONSTANTS[vmyo]);
RATES[cass] =  ALGEBRAIC[Bcass]*((( - (ALGEBRAIC[ICaL] -  2.00000*ALGEBRAIC[INaCa_ss])*CONSTANTS[cm]*CONSTANTS[Acap])/( 2.00000*CONSTANTS[F]*CONSTANTS[vss])+( ALGEBRAIC[Jrel]*CONSTANTS[vjsr])/CONSTANTS[vss]) - ALGEBRAIC[Jdiff]);
RATES[cansr] = ALGEBRAIC[Jup] - ( ALGEBRAIC[Jtr]*CONSTANTS[vjsr])/CONSTANTS[vnsr];
RATES[cajsr] =  ALGEBRAIC[Bcajsr]*(ALGEBRAIC[Jtr] - ALGEBRAIC[Jrel]);
RATES[V] = - (ALGEBRAIC[INa]+ALGEBRAIC[INaL]+ALGEBRAIC[Ito]+ALGEBRAIC[ICaL]+ALGEBRAIC[ICaNa]+ALGEBRAIC[ICaK]+ALGEBRAIC[IKr]+ALGEBRAIC[IKs]+ALGEBRAIC[IK1]+ALGEBRAIC[INaCa_i]+ALGEBRAIC[INaCa_ss]+ALGEBRAIC[INaK]+ALGEBRAIC[INab]+ALGEBRAIC[IKb]+ALGEBRAIC[IpCa]+ALGEBRAIC[ICab]+ALGEBRAIC[Istim]);
}



drug_t ic50;
__device__ drug_t *d_ic50;
// double ic50[2000][14];
// double *d_ic50[2000][14];

double *d_concs[4];
__device__ double *d_time_step;

// __global__ void toc(clock_t start = START_TIMER);

__global__ void check_data(){
  printf("check data: \n");
  int idx = 14;
  for(int sample_index=0; sample_index<idx; sample_index++){
        printf("%lf|", d_ic50[2][sample_index]);
        }
     //   printf("\n \n");
}

__global__ void set_time_step(
  /*
  as 'adaptive' solver, we need the time step to change in the middle of 
  the process
  since we need to change almost every function to void, I change the 
  return time_step to 
  cudaMemCopy the time_step, 
  */
    double TIME,
    double time_point,
    double max_time_step,
    double* CONSTANTS,
    double* RATES,
    double* STATES,
    double* ALGEBRAIC) {
    double time_step = 0.005;

    if (TIME <= time_point || (TIME - floor(TIME / CONSTANTS[stim_period]) * CONSTANTS[stim_period]) <= time_point) {
        //printf("TIME <= time_point ms\n");
        //return time_step;
        memcpy(d_time_step, &time_step, sizeof(double));
        __syncthreads(); //equivalent to break
        //printf("dV = %lf, time_step = %lf\n",RATES[V] * time_step, time_step);
    }
    else {
        //printf("TIME > time_point ms\n");
        if (std::abs(RATES[V] * time_step) <= 0.2) {//Slow changes in V
            //printf("dV/dt <= 0.2\n");
            time_step = std::abs(0.8 / RATES[V]);
            //Make sure time_step is between 0.005 and max_time_step
            if (time_step < 0.005) {
                time_step = 0.005;
            }
            else if (time_step > max_time_step) {
                time_step = max_time_step;
            }
            //printf("dV = %lf, time_step = %lf\n",std::abs(RATES[V] * time_step), time_step);
        }
        else if (std::abs(RATES[V] * time_step) >= 0.8) {//Fast changes in V
            //printf("dV/dt >= 0.8\n");
            time_step = std::abs(0.2 / RATES[V]);
            while (std::abs(RATES[V] * time_step) >= 0.8 && 0.005 < time_step && time_step < max_time_step) {
                time_step = time_step / 10.0;
                //printf("dV = %lf, time_step = %lf\n",std::abs(RATES[V] * time_step), time_step);
            }
        }
        // return time_step;
        memcpy(d_time_step, &time_step, sizeof(double));
    }
}
/*__global__ void solveAnalytical(double dt)
{ 
  ////==============
  ////Exact solution
  ////==============
  ////INa
  STATES[m] = ALGEBRAIC[mss] - (ALGEBRAIC[mss] - STATES[m]) * exp(-dt / ALGEBRAIC[tm]);
  STATES[hf] = ALGEBRAIC[hss] - (ALGEBRAIC[hss] - STATES[hf]) * exp(-dt / ALGEBRAIC[thf]);
  STATES[hs] = ALGEBRAIC[hss] - (ALGEBRAIC[hss] - STATES[hs]) * exp(-dt / ALGEBRAIC[ths]);
  STATES[j] = ALGEBRAIC[jss] - (ALGEBRAIC[jss] - STATES[j]) * exp(-dt / ALGEBRAIC[tj]);
  STATES[hsp] = ALGEBRAIC[hssp] - (ALGEBRAIC[hssp] - STATES[hsp]) * exp(-dt / ALGEBRAIC[thsp]);
  STATES[jp] = ALGEBRAIC[jss] - (ALGEBRAIC[jss] - STATES[jp]) * exp(-dt / ALGEBRAIC[tjp]);
  STATES[mL] = ALGEBRAIC[mLss] - (ALGEBRAIC[mLss] - STATES[mL]) * exp(-dt / ALGEBRAIC[tmL]);
  STATES[hL] = ALGEBRAIC[hLss] - (ALGEBRAIC[hLss] - STATES[hL]) * exp(-dt / CONSTANTS[thL]);
  STATES[hLp] = ALGEBRAIC[hLssp] - (ALGEBRAIC[hLssp] - STATES[hLp]) * exp(-dt / CONSTANTS[thLp]);
  ////Ito
  STATES[a] = ALGEBRAIC[ass] - (ALGEBRAIC[ass] - STATES[a]) * exp(-dt / ALGEBRAIC[ta]);
  STATES[iF] = ALGEBRAIC[iss] - (ALGEBRAIC[iss] - STATES[iF]) * exp(-dt / ALGEBRAIC[tiF]);
  STATES[iS] = ALGEBRAIC[iss] - (ALGEBRAIC[iss] - STATES[iS]) * exp(-dt / ALGEBRAIC[tiS]);
  STATES[ap] = ALGEBRAIC[assp] - (ALGEBRAIC[assp] - STATES[ap]) * exp(-dt / ALGEBRAIC[ta]);
  STATES[iFp] = ALGEBRAIC[iss] - (ALGEBRAIC[iss] - STATES[iFp]) * exp(-dt / ALGEBRAIC[tiFp]);
  STATES[iSp] = ALGEBRAIC[iss] - (ALGEBRAIC[iss] - STATES[iSp]) * exp(-dt / ALGEBRAIC[tiSp]);
  ////ICaL
  STATES[d] = ALGEBRAIC[dss] - (ALGEBRAIC[dss] - STATES[d]) * exp(-dt / ALGEBRAIC[td]);
  STATES[ff] = ALGEBRAIC[fss] - (ALGEBRAIC[fss] - STATES[ff]) * exp(-dt / ALGEBRAIC[tff]);
  STATES[fs] = ALGEBRAIC[fss] - (ALGEBRAIC[fss] - STATES[fs]) * exp(-dt / ALGEBRAIC[tfs]);
  STATES[fcaf] = ALGEBRAIC[fcass] - (ALGEBRAIC[fcass] - STATES[fcaf]) * exp(-dt / ALGEBRAIC[tfcaf]);
  STATES[fcas] = ALGEBRAIC[fcass] - (ALGEBRAIC[fcass] - STATES[fcas]) * exp(-dt / ALGEBRAIC[tfcas]);
  STATES[jca] = ALGEBRAIC[fcass] - (ALGEBRAIC[fcass] - STATES[jca]) * exp(- dt / CONSTANTS[tjca]);
  STATES[ffp] = ALGEBRAIC[fss] - (ALGEBRAIC[fss] - STATES[ffp]) * exp(-dt / ALGEBRAIC[tffp]);
  STATES[fcafp] = ALGEBRAIC[fcass] - (ALGEBRAIC[fcass] - STATES[fcafp]) * exp(-d / ALGEBRAIC[tfcafp]);
  STATES[nca] = ALGEBRAIC[anca] * CONSTANTS[k2n] / ALGEBRAIC[km2n] -
      (ALGEBRAIC[anca] * CONSTANTS[k2n] / ALGEBRAIC[km2n] - STATES[nca]) * exp(-ALGEBRAIC[km2n] * dt);
  ////IKr
  STATES[xrf] = ALGEBRAIC[xrss] - (ALGEBRAIC[xrss] - STATES[xrf]) * exp(-dt / ALGEBRAIC[txrf]);
  STATES[xrs] = ALGEBRAIC[xrss] - (ALGEBRAIC[xrss] - STATES[xrs]) * exp(-dt / ALGEBRAIC[txrs]);
  ////IKs
  STATES[xs1] = ALGEBRAIC[xs1ss] - (ALGEBRAIC[xs1ss] - STATES[xs1]) * exp(-dt / ALGEBRAIC[txs1]);
  STATES[xs2] = ALGEBRAIC[xs2ss] - (ALGEBRAIC[xs2ss] - STATES[xs2]) * exp(-dt / ALGEBRAIC[txs2]);
  ////IK1
  STATES[xk1] = ALGEBRAIC[xk1ss] - (ALGEBRAIC[xk1ss] - STATES[xk1]) * exp(-dt / ALGEBRAIC[txk1]);
  ////INaCa
  ////INaK
  ////IKb
  ////INab
  ////ICab
  ///IpCa
  ////Diffusion fluxes
  ////RyR receptors
  STATES[Jrelnp] = ALGEBRAIC[Jrel_inf] - (ALGEBRAIC[Jrel_inf] - STATES[Jrelnp]) * exp(-dt / ALGEBRAIC[tau_rel]);
  STATES[Jrelp] = ALGEBRAIC[Jrel_infp] - (ALGEBRAIC[Jrel_infp] - STATES[Jrelp]) * exp(-dt / ALGEBRAIC[tau_relp]);
  ////SERCA Pump
  ////Calcium translocation
  //
  ////=============================
  ////Approximated solution (Euler)
  ////=============================
  ////ICaL
  //STATES[jca] = STATES[jca] + RATES[jca] * dt;
  ////CaMK
  STATES[CaMKt] = STATES[CaMKt] + RATES[CaMKt] * dt;
  ////Membrane potential
  STATES[V] = STATES[V] + RATES[V] * dt;
  ////Ion Concentrations and Buffers
  STATES[nai] = STATES[nai] + RATES[nai] * dt;
  STATES[nass] = STATES[nass] + RATES[nass] * dt;
  STATES[ki] = STATES[ki] + RATES[ki] * dt;
  STATES[kss] = STATES[kss] + RATES[kss] * dt;
  STATES[cai] = STATES[cai] + RATES[cai] * dt;
  STATES[cass] = STATES[cass] + RATES[cass] * dt;
  STATES[cansr] = STATES[cansr] + RATES[cansr] * dt;
  STATES[cajsr] = STATES[cajsr] + RATES[cajsr] * dt; 
  //========================
  //Full Euler Approximation
  //========================
  //STATES[V] = STATES[V] + RATES[V] * dt;
  //STATES[CaMKt] = STATES[CaMKt] + RATES[CaMKt] * dt;
  //STATES[cass] = STATES[cass] + RATES[cass] * dt;
  //STATES[nai] = STATES[nai] + RATES[nai] * dt;
  //STATES[nass] = STATES[nass] + RATES[nass] * dt;
  //STATES[ki] = STATES[ki] + RATES[ki] * dt;
  //STATES[kss] = STATES[kss] + RATES[kss] * dt;
  //STATES[cansr] = STATES[cansr] + RATES[cansr] * dt;
  //STATES[cajsr] = STATES[cajsr] + RATES[cajsr] * dt;
  //STATES[cai] = STATES[cai] + RATES[cai] * dt;
  //STATES[m] = STATES[m] + RATES[m] * dt;
  //STATES[hf] = STATES[hf] + RATES[hf] * dt;
  //STATES[hs] = STATES[hs] + RATES[hs] * dt;
  //STATES[j] = STATES[j] + RATES[j] * dt;
  //STATES[hsp] = STATES[hsp] + RATES[hsp] * dt;
  //STATES[jp] = STATES[jp] + RATES[jp] * dt;
  //STATES[mL] = STATES[mL] + RATES[mL] * dt;
  //STATES[hL] = STATES[hL] + RATES[hL] * dt;
  //STATES[hLp] = STATES[hLp] + RATES[hLp] * dt;
  //STATES[a] = STATES[a] + RATES[a] * dt;
  //STATES[iF] = STATES[iF] + RATES[iF] * dt;
  //STATES[iS] = STATES[iS] + RATES[iS] * dt;
  //STATES[ap] = STATES[ap] + RATES[ap] * dt;
  //STATES[iFp] = STATES[iFp] + RATES[iFp] * dt;
  //STATES[iSp] = STATES[iSp] + RATES[iSp] * dt;
  //STATES[d] = STATES[d] + RATES[d] * dt;
  //STATES[ff] = STATES[ff] + RATES[ff] * dt;
  //STATES[fs] = STATES[fs] + RATES[fs] * dt;
  //STATES[fcaf] = STATES[fcaf] + RATES[fcaf] * dt;
  //STATES[fcas] = STATES[fcas] + RATES[fcas] * dt;
  //STATES[jca] = STATES[jca] + RATES[jca] * dt;
  //STATES[ffp] = STATES[ffp] + RATES[ffp] * dt;
  //STATES[fcafp] = STATES[fcafp] + RATES[fcafp] * dt;
  //STATES[nca] = STATES[nca] + RATES[nca] * dt;
  //STATES[xrf] = STATES[xrf] + RATES[xrf] * dt;
  //STATES[xrs] = STATES[xrs] + RATES[xrs] * dt;
  //STATES[xs1] = STATES[xs1] + RATES[xs1] * dt;
  //STATES[xs2] = STATES[xs2] + RATES[xs2] * dt;
  //STATES[xk1] = STATES[xk1] + RATES[xk1] * dt;
  //STATES[Jrelnp] = STATES[Jrelnp] + RATES[Jrelnp] * dt;
  //STATES[Jrelp] = STATES[Jrelp] + RATES[Jrelp] * dt;
}
*/


__global__ void do_drug_sim_analytical(double conc, drug_t *d_ic50, const param_t* p_param, 
const unsigned short sample_id)
{

  /*
  do drug effect simulation, loop will be replaced with kernel loops
  */
  double tcurr = 0.0, dt = 0.005, dt_set, tmax;
  double max_time_step = 1.0, time_point = 25.0;
  
  // files for storing results
  // time-series result
  FILE *vfp_m, *fp_inet, *fp_gate;

  // features
  double inet, qnet;

  // looping counter
  unsigned short idx = 14;
  
  // simulation parameters
  double dtw = 2.0;
  const char *drug_name = "bepridil";
  const double bcl = 2000;
  const double inet_vm_threshold = -88.0;
  const unsigned short pace_max = 10;
  const unsigned short celltype = 0.;
  const unsigned short last_pace_print = 3;
  const unsigned short last_drug_check_pace = 250;
  const unsigned int print_freq = (1./dt) * dtw;
  unsigned short pace_count = 0;
  unsigned short pace_steepest = 0;

  double* RATES;
  double* STATES;
  double* CONSTANTS;
  double* ALGEBRAIC;

  int num_of_algebraic = 69;
  int num_of_constants = 46;
  int num_of_rates = 17;
  int num_of_states = 17;
  //hipMalloc((drug_t**)&d_ic50, sizeof(drug_t));
  RATES = (double*)malloc((num_of_rates)*sizeof(double));
  STATES = (double*)malloc((num_of_states)*sizeof(double));
  CONSTANTS = (double*)malloc((num_of_constants)*sizeof(double));
  ALGEBRAIC = (double*)malloc((num_of_algebraic)*sizeof(double));

  // apply some cell initialization
  initConsts<<<1,1>>>(CONSTANTS, RATES, STATES);
  printf("constants: %lf rates: %lf states: %lf \n",CONSTANTS,RATES,STATES);
  //p_cell->initConsts( celltype, conc, ic50.data());
  CONSTANTS[stim_period] = bcl;

  // generate file for time-series output
  // snprintf(buffer, sizeof(buffer), "result/%s_%.2lf_vmcheck_smp%d.plt", 
  //           drug_name, conc, sample_id );
  // fp_vm = fopen( buffer, "w" );
  // snprintf(buffer, sizeof(buffer), "result/%s_%.2lf_gates_smp%d.plt",
  //           drug_name, conc, sample_id);
  // fp_gate = fopen(buffer, "w");
  // printf("drug name: %s , concentration: %.2lf , sample id: %d \n", drug_name, conc, sample_id);
  printf("\n");

  // printf(fp_vm, "%s %s\n", "Time", "Vm");
  //printf("Time: %s Vm: %s\n", "Time", "Vm");
  // fprintf(fp_gate, "Time %s\n", GATES_HEADER); //this is to write headers in results

  tmax = pace_max * bcl;

  while (tcurr < tmax) {
    // dt_set = set_time_step<<<1,1>>>(tcurr,
    //     		   time_point,
		//            max_time_step,
  	// 	         CONSTANTS,
		//            RATES,
		// 	         STATES,
		//            ALGEBRAIC);
    // set_time_step<<<1,1>>>(tcurr,
    //     		   time_point,
		//            max_time_step,
  	// 	         CONSTANTS,
		//            RATES,
		// 	         STATES,
		//            ALGEBRAIC);
              // hipDeviceSynchronize();
    // printf("set time step\n");
    //printf("timestep pointer: %x \n",d_time_step);
    //dt_set = *d_time_step;
    dt_set = 0.0001;

    // // //Compute all rates at tcurr
    // computeRates<<<1,1>>>(tcurr,
		//           CONSTANTS,
    //         	RATES,
		//           STATES,
    //         	ALGEBRAIC);
              // hipDeviceSynchronize();
    // printf("compute rates at tcurr\n");

    //Compute the correct/accepted time step
    if (floor((tcurr + dt_set) / bcl) == floor(tcurr / bcl)) {
      dt = dt_set;
    }
    else {
      dt = (floor(tcurr / bcl) + 1) * bcl - tcurr;
    }

    //Compute the analytical solution
    //solveAnalytical<<<1,1>>>(dt);
    //printf("solve analytical done\n");
    
    //=============//
    //Print results//
    //=============//
    // fprintf(fp_vm, "%lf %lf\n", tcurr, STATES[V]);
    // fprintf(fp_gate, "%lf ",tcurr);
    //printf("tcurr: %lf States[V]: %lf\n", tcurr, STATES[V]);
    // printf("%lf \n \n",tcurr);    
    // for(idx = 0; idx < p_cell->gates_size; idx++){
    //   fprintf(fp_gate, "%lf ", p_cell->STATES[p_cell->GATES_INDICES[idx]]);
    // }
    // fprintf(fp_gate, "\n");
    printf("\n");
    
    //Next time step
    tcurr = tcurr + dt;
  }

  // clean the memories
  //fclose(fp_vm);
  //fclose(fp_gate);
}


//__global__ void Calculate(double d_ic50[11][14], double concs[4], Cellmodel *p_cell);
__global__ void Concentration(drug_t *d_ic50, double *concs[4]){
  
  /*
  uses block and thread in CUDA to replace concentration loop
  */

  // Get the thread ID.
  int sample_id = threadIdx.x;
  int conc_idx = blockIdx.x;
  //printf("doing calculation loop....\n");
  

  //for now, we hard code the concs
  double h_concs[4] = {0.0, 33.0, 66.0, 99.0};

  //memset(h_concs, -1, sizeof(h_concs));
  //printf("%lf", h_concs[1]);
  // hipMemcpy(d_p_cell, p_cell, sizeof(Cellmodel), hipMemcpyHostToDevice);
  // hipMemcpy(h_concs, concs, 4*sizeof(double), hipMemcpyDeviceToHost);

  // printf("concentration: %d -> value: %lf\n",conc_idx, h_concs[conc_idx]);
  // printf("Sample_ID: %d\n",sample_id );
  
  
  //       printf("\n");
        // for( const auto &conc: concs )
        // { // begin concentration loop
        // printf("Current Concentration: %lf  ",concs[a]);
        // // execute main simulation function
        // //do_drug_sim(conc, ic50[sample_id],
        // //            NULL, sample_id,
        // //            p_cell, ode_solver, cvode_firsttime);
        // // TODO @IritaSee: paralelise this loop that takes each data 
        
        //WARNING: concs still hard coded
       //do_drug_sim_analytical<<<1,1>>>(h_concs[conc_idx], *d_ic50[sample_id], NULL, sample_id);
       do_drug_sim_analytical<<<1,1>>>(h_concs[conc_idx], d_ic50, NULL, sample_id);

        // } // end concentration loop

}


int main()
{

    // input variables for cell simulation
    double bcl, dt;
    unsigned short pace;

    //prepare memory slots for ic_50 
    hipSetDevice(0);
    hipMalloc((drug_t**)&d_ic50, sizeof(drug_t));
    //perpare memory slots for concentration and copy it to the just created mem slots
    hipMalloc((void**)&d_concs, 4*sizeof(double)); 
    hipMemcpy(d_concs, concs, 4*sizeof(double), hipMemcpyHostToDevice);
    //prepare memory slots for p_cell and copy it
    // hipMalloc((void**)d_p_cell, sizeof(Cellmodel));
    // hipMemcpy(d_p_cell, p_cell, sizeof(Cellmodel), hipMemcpyHostToDevice);
    unsigned short idx;
    tic();
    snprintf(buffer, sizeof(buffer),
      "./drugs/bepridil/IC50_samples10.csv");
    //drug_t ic50 = get_IC50_data_from_file(buffer);
    //int data_row = sizeof(ic50)/sizeof(ic50[0]);
    int data_row = 10;
    get_IC50_data_from_file(buffer);
    if(sizeof(ic50)/sizeof(ic50[0]) == 0)
        printf("Something problem with the IC50 file!\n");
    else if(sizeof(ic50)/sizeof(ic50[0]) > 2000)
        printf("Too much input! Maximum sample data is 2000!\n");
    printf("start calculation....\n");
    // dim3 block(32,32);
    //dim3 grid ((columns+block.x-1)/block.x,(rows+block.y-1)/block.y);
    Concentration<<<4,data_row>>>(d_ic50, d_concs );  
    // Calculate(d_ic50, d_concs, d_p_cell );
    //concentration loop fails so i loop it altogether
    hipDeviceSynchronize();
    toc(START_TIMER);
    // loop to do calculation in each data is replaced by this func
    
    // memory cleaning and finalize the program
    

    return 0;
}

void get_IC50_data_from_file(const char* file_name)
{
  /*get IC50 data from a file*/
  /*caution: keep it host function!*/
  FILE *fp_drugs;
  printf("Reading the data....\n");
  
  char *token;
  //std::array<double,14> temp_array; //make the d_ version as well?
  double temp_array[1][14];
  //unsigned short idx;
  unsigned int idx;

  if( (fp_drugs = fopen(file_name, "r")) == NULL){
    printf("Cannot open file %s\n",
      file_name);
    //return ic50;
  }

  int count = 0;

  fgets(buffer, sizeof(buffer), fp_drugs); // skip header
  while( fgets(buffer, sizeof(buffer), fp_drugs) != NULL )
  { // begin line reading
    token = strtok( buffer, "," );
    idx = 0;
    while( token != NULL )
    { // begin data tokenizing
      temp_array[0][idx] = strtod(token, NULL);
      token = strtok(NULL, ",");
      ic50[count][idx] = temp_array[0][idx];
      idx=idx+1;
    } // end data tokenizing
    for(int sample_index=0; sample_index<idx; sample_index++){
        printf("%lf|", ic50[count][sample_index]);
        }
        printf("\n \n");
    //ic50.push_back(temp_array);
    count = count+1;
  } // end line reading

  fclose(fp_drugs);

  //copy the ic50 to GPU memory
  printf("rows found: %d\n",idx);
  
  hipMemcpy(d_ic50, ic50, idx * sizeof(drug_t), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  check_data<<<1,1>>>();
  //  printf("device memory sample contents: ");
  //  for(int sample_index=0; sample_index<idx; sample_index++){
  //       printf("%lf|", *d_ic50[1][sample_index]);
  //       }
  //       printf("\n \n");

  //return ic50;
}
