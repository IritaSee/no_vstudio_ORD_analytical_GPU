#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 10

__global__ void fun(){
    printf("%d\n", threadIdx.x);
    }

int main(){
    fun<<<1,N>>>();
    hipDeviceSynchronize();
    return 0;
}