#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

using namespace std;

__global__ int devAr[1];

__global__ void foo()
{
    printf("\n");
    printf("dev before init: %d \n", devAr[0]);
    devAr[0] = 77;
    printf("dev after init: %d \n", devAr[0]);
    printf("\n");
    // printf("cek: %d \n", cek);
}

int main()
{
    int test = 10;
    cout << "host initially: " << devAr[0] << endl;
    devAr[0] = 4;
    cout << "host after init: " << devAr[0] << endl;
    foo << <1, 1 >> >() ;
    hipDeviceSynchronize();
    cout << "host after foo: " << devAr[0] << endl;
}