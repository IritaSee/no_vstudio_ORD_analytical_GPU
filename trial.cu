#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(){
    int t = threadIdx.x;
    int T = blockIdx.x;
    printf("tread index: %d  block index: %d\n",t,T);
}
 // compile like usual c code (nvcc trial.cu -o trial) will create trial.exe
int main(){
    //syntax:
    //[global or host void name]<<<grid_dimension, block_dimension>>>();
    int block_dim = 10; // how many threads are in one block
    int grid_dim = 3; // hown many blocks are in one grid
    dkernel<<<grid_dim,block_dim>>>(); //these are called kernels -> when called, are executed 32 times in parallel by 32 different CUDA threads, as opposed to only once like regular C functions.
    /*
     configuration syntax (see C Language Extensions). Each thread that executes the kernel is given a unique 
    thread ID that is accessible within the kernel through the built-in threadIdx variable.
    */
    hipDeviceSynchronize();
    /*
    The CUDA code you have provided is syntactically correct. 
    However, it will not print anything to the console. 
    This is because the kernel is not actually running on the GPU. 
    To run the kernel on the GPU, you need to call the cudaDeviceSynchronize() function. 
    This function will ensure that the kernel has finished executing before the main() function returns.
    */
    //cudaThreadSynchronize(); -> deprecated
    return 0;
}
/*
There are 4 variables each thread can access which contains information 
about the organization of threads and current thread. They are:
threadIdx : Id of the current thread.
blockIdx : Id of the current block.
blockDim : Size of each dimension of the current block.
gridDim : Size of each dimension of the current grid.
All of these are dim3 structure. we can use dot notation to access variable 
x,y,z which contains the information of the corresponding dimension. Example: threadIdx.x
*/
