#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(){
    printf("Hello Cuda\n");
}
 // compile like usual c code (nvcc trial.cu -o trial) will create trial.exe
int main(){
    //syntax:
    //[global or host void name]<<<kernel invocation, kernel threads>>>();
    dkernel<<<100,1000>>>(); //these are called kernels -> when called, are executed 32 times in parallel by 32 different CUDA threads, as opposed to only once like regular C functions.
    /*
    A kernel is defined using the __global__ declaration specifier and the number of CUDA threads 
    that execute that kernel for a given kernel call is specified using a new <<<...>>> execution
    configuration syntax (see C Language Extensions). Each thread that executes the kernel is given a unique 
    thread ID that is accessible within the kernel through the built-in threadIdx variable.
    */
    hipDeviceSynchronize();
    return 0;
}