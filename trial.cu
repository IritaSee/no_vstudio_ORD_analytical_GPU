#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(){
    int t = threadIdx.x;
    int T = blockIdx.x;
    printf(" tread index: %d  block index: %d\n",t,T);
}
 // compile like usual c code (nvcc trial.cu -o trial) will create trial.exe
int main(){
    //syntax:
    //[global or host void name]<<<block, thread per block>>>();
    dkernel<<<3,10>>>(); //these are called kernels -> when called, are executed 32 times in parallel by 32 different CUDA threads, as opposed to only once like regular C functions.
    /*
    A kernel is defined using the __global__ declaration specifier and the number of CUDA threads 
    that execute that kernel for a given kernel call is specified using a new <<<...>>> execution
    configuration syntax (see C Language Extensions). Each thread that executes the kernel is given a unique 
    thread ID that is accessible within the kernel through the built-in threadIdx variable.
    */
    hipDeviceSynchronize();
    /*
    The CUDA code you have provided is syntactically correct. 
    However, it will not print anything to the console. 
    This is because the kernel is not actually running on the GPU. 
    To run the kernel on the GPU, you need to call the cudaDeviceSynchronize() function. 
    This function will ensure that the kernel has finished executing before the main() function returns.
    */
    //cudaThreadSynchronize(); -> deprecated
    return 0;
/*
There are 4 variables each thread can access which contains information 
about the organization of threads and current thread. They are:

threadIdx : Id of the current thread.
blockIdx : Id of the current block.
blockDim : Size of each dimension of the current block.
gridDim : Size of each dimension of the current grid.

All of these are dim3 structure. we can use dot notation to access variable 
x,y,z which contains the information of the corresponding dimension. Example: threadIdx.x
*/

}