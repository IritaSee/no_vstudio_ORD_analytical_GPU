#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(){
    printf("Hello Cuda");
}
 // compile like usual c code (nvcc trial.cu -o trial) will create trial.exe
int main(){
    dkernel<<<1,1>>>();
    return 0;
}