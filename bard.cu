#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void multiply(int *a, int *b, int *c, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    c[i] = a[i] * b[i];
  }
}

int main() {
  // Declare the input and output arrays.
  int *a = (int *)malloc(sizeof(int) * 10);
  int *b = (int *)malloc(sizeof(int) * 10);
  int *c = (int *)malloc(sizeof(int) * 10);

  // Initialize the input arrays.
  for (int i = 0; i < 10; i++) {
    a[i] = i;
    b[i] = i + 1;
  }

  // Check for errors when copying the input arrays to the GPU.
  hipError_t cudaStatus = hipMemcpy(a, a, sizeof(int) * 10, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "CUDA memcpy failed: %s\n", hipGetErrorString(cudaStatus));
    return 1;
  }

  // Check for errors when copying the input arrays to the GPU.
  cudaStatus = hipMemcpy(b, b, sizeof(int) * 10, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "CUDA memcpy failed: %s\n", hipGetErrorString(cudaStatus));
    return 1;
  }

  // Execute the kernel function.
  dim3 blockDim(1024, 1);
  dim3 gridDim(1, 1);
  multiply<<<gridDim, blockDim>>>(a, b, c, 10);

  // Check for errors when copying the output array from the GPU.
  cudaStatus = hipMemcpy(c, c, sizeof(int) * 10, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "CUDA memcpy failed: %s\n", hipGetErrorString(cudaStatus));
    return 1;
  }

  // Print the output array.
  for (int i = 0; i < 10; i++) {
    printf("%d * %d = %d\n",a[i],b[i],c[i]);
  }

  // Free the memory allocated for the input and output arrays.
  free(a);
  free(b);
  free(c);

  return 0;
}
