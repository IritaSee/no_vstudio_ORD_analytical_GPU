#include "hip/hip_runtime.h"
#include <iostream>

 __global__ void child_kernel() {printf("hello\n");}

 __global__ void addKernel(int *c, const int *a, const int *b)
 {
     child_kernel << <1, 1 >> > ();
     int i = threadIdx.x;
     c[i] = a[i] + b[i];
 }

 int main (){
    int c[5];
    int a[5] = {1,2,3,4,5};
    int b[5] = {2,3,4,5,6};
    addKernel<<<1,1>>>(c,a,b);
    hipDeviceSynchronize();
 }