#include "hip/hip_runtime.h"
// CVodeSimTestSimple.cpp : This file contains the 'main' function. Program execution begins and ends there.
//
#include <array>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstring>
#include <ctime>
#include <iostream>

#include <hip/hip_runtime.h>

#include "enums/enum_mar_cell_MKII.cuh"

#include "modules/globals.hpp"
#include "modules/commons.hpp"


clock_t START_TIMER;

char buffer[255];
double concs [4] = {0.0, 33.0, 66.0, 99.0};
// variables for I/O
FILE* fp_vm;
FILE* fp_gate;
    
// timer section
clock_t tic()
/* start timer*/
{
    return START_TIMER = clock();
}

void toc(clock_t start)
{
  /*stop timer*/
    std::cout
        << "Elapsed time: "
        << (clock() - start) / (double)CLOCKS_PER_SEC << "s"
        << std::endl;
}


void get_IC50_data_from_file(const char* file_name);

__global__ void initConsts(double* CONSTANTS, double* RATES, double *STATES){
    STATES[0] = -86.2;
CONSTANTS[0] = 8.314;
CONSTANTS[1] = 310;
CONSTANTS[2] = 96.485;
CONSTANTS[3] = 185;
CONSTANTS[4] = 16404;
CONSTANTS[5] = 10;
CONSTANTS[6] = 1000;
CONSTANTS[7] = 1;
CONSTANTS[8] = -52;
CONSTANTS[9] = 0.03;
CONSTANTS[10] = 5.4;
CONSTANTS[11] = 140;
STATES[1] = 138.3;
STATES[2] = 11.6;
CONSTANTS[12] = 2;
STATES[3] = 0.0002;
CONSTANTS[13] = 5.405;
CONSTANTS[14] = 0.096;
STATES[4] = 0;
STATES[5] = 1;
CONSTANTS[15] = 0.062;
STATES[6] = 0;
CONSTANTS[16] = 14.838;
STATES[7] = 0;
STATES[8] = 0.75;
STATES[9] = 0.75;
CONSTANTS[17] = 0.00029;
CONSTANTS[18] = 0.175;
STATES[10] = 0;
STATES[11] = 1;
STATES[12] = 1;
CONSTANTS[19] = 0.000592;
CONSTANTS[20] = 0.294;
STATES[13] = 1;
STATES[14] = 0;
CONSTANTS[21] = 1.362;
CONSTANTS[22] = 1;
CONSTANTS[23] = 40;
CONSTANTS[24] = 1000;
CONSTANTS[25] = 0.1;
CONSTANTS[26] = 2.5;
CONSTANTS[27] = 0.35;
CONSTANTS[28] = 1.38;
CONSTANTS[29] = 87.5;
CONSTANTS[30] = 0.825;
CONSTANTS[31] = 0.0005;
CONSTANTS[32] = 0.0146;
STATES[15] = 0.2;
STATES[16] = 1;
CONSTANTS[33] = 2;
CONSTANTS[34] = 0.016464;
CONSTANTS[35] = 0.25;
CONSTANTS[36] = 0.008232;
CONSTANTS[37] = 0.00025;
CONSTANTS[38] = 8e-5;
CONSTANTS[39] = 0.000425;
CONSTANTS[40] = 0.15;
CONSTANTS[41] = 0.001;
CONSTANTS[42] = 10;
CONSTANTS[43] = 0.3;
CONSTANTS[44] = 1094;
CONSTANTS[45] = 2.00000;
}

__global__ void computeRates(double VOI, double* CONSTANTS, double* RATES, double* STATES, double* ALGEBRAIC)
{
ALGEBRAIC[8] = 1.00000/(1.00000+exp((STATES[0]+20.0000)/7.00000));
ALGEBRAIC[21] =  1125.00*exp(- pow(STATES[0]+27.0000, 2.00000)/240.000)+80.0000+165.000/(1.00000+exp((25.0000 - STATES[0])/10.0000));
RATES[11] = (ALGEBRAIC[8] - STATES[11])/ALGEBRAIC[21];
ALGEBRAIC[10] = 1.00000/(1.00000+exp((STATES[0]+20.0000)/5.00000));
ALGEBRAIC[23] =  85.0000*exp(- pow(STATES[0]+45.0000, 2.00000)/320.000)+5.00000/(1.00000+exp((STATES[0] - 20.0000)/5.00000))+3.00000;
RATES[13] = (ALGEBRAIC[10] - STATES[13])/ALGEBRAIC[23];
ALGEBRAIC[11] = 1.00000/(1.00000+exp((20.0000 - STATES[0])/6.00000));
ALGEBRAIC[24] =  9.50000*exp(- pow(STATES[0]+40.0000, 2.00000)/1800.00)+0.800000;
RATES[14] = (ALGEBRAIC[11] - STATES[14])/ALGEBRAIC[24];
ALGEBRAIC[12] = (STATES[3]<0.000350000 ? 1.00000/(1.00000+pow(STATES[3]/0.000350000, 6.00000)) : 1.00000/(1.00000+pow(STATES[3]/0.000350000, 16.0000)));
ALGEBRAIC[25] = (ALGEBRAIC[12] - STATES[16])/CONSTANTS[33];
RATES[16] = (ALGEBRAIC[12]>STATES[16]&&STATES[0]>- 60.0000 ? 0.00000 : ALGEBRAIC[25]);
ALGEBRAIC[1] = 1.00000/(1.00000+exp((- 26.0000 - STATES[0])/7.00000));
ALGEBRAIC[14] = 450.000/(1.00000+exp((- 45.0000 - STATES[0])/10.0000));
ALGEBRAIC[27] = 6.00000/(1.00000+exp((STATES[0]+30.0000)/11.5000));
ALGEBRAIC[36] =  1.00000*ALGEBRAIC[14]*ALGEBRAIC[27];
RATES[4] = (ALGEBRAIC[1] - STATES[4])/ALGEBRAIC[36];
ALGEBRAIC[2] = 1.00000/(1.00000+exp((STATES[0]+88.0000)/24.0000));
ALGEBRAIC[15] = 3.00000/(1.00000+exp((- 60.0000 - STATES[0])/20.0000));
ALGEBRAIC[28] = 1.12000/(1.00000+exp((STATES[0] - 60.0000)/20.0000));
ALGEBRAIC[37] =  1.00000*ALGEBRAIC[15]*ALGEBRAIC[28];
RATES[5] = (ALGEBRAIC[2] - STATES[5])/ALGEBRAIC[37];
ALGEBRAIC[3] = 1.00000/(1.00000+exp((- 5.00000 - STATES[0])/14.0000));
ALGEBRAIC[16] = 1100.00/ pow((1.00000+exp((- 10.0000 - STATES[0])/6.00000)), 1.0 / 2);
ALGEBRAIC[29] = 1.00000/(1.00000+exp((STATES[0] - 60.0000)/20.0000));
ALGEBRAIC[38] =  1.00000*ALGEBRAIC[16]*ALGEBRAIC[29];
RATES[6] = (ALGEBRAIC[3] - STATES[6])/ALGEBRAIC[38];
ALGEBRAIC[4] = 1.00000/pow(1.00000+exp((- 56.8600 - STATES[0])/9.03000), 2.00000);
ALGEBRAIC[17] = 1.00000/(1.00000+exp((- 60.0000 - STATES[0])/5.00000));
ALGEBRAIC[30] = 0.100000/(1.00000+exp((STATES[0]+35.0000)/5.00000))+0.100000/(1.00000+exp((STATES[0] - 50.0000)/200.000));
ALGEBRAIC[39] =  1.00000*ALGEBRAIC[17]*ALGEBRAIC[30];
RATES[7] = (ALGEBRAIC[4] - STATES[7])/ALGEBRAIC[39];
ALGEBRAIC[5] = 1.00000/pow(1.00000+exp((STATES[0]+71.5500)/7.43000), 2.00000);
ALGEBRAIC[18] = (STATES[0]<- 40.0000 ?  0.0570000*exp(- (STATES[0]+80.0000)/6.80000) : 0.00000);
ALGEBRAIC[31] = (STATES[0]<- 40.0000 ?  2.70000*exp( 0.0790000*STATES[0])+ 310000.*exp( 0.348500*STATES[0]) : 0.770000/( 0.130000*(1.00000+exp((STATES[0]+10.6600)/- 11.1000))));
ALGEBRAIC[40] = 1.00000/(ALGEBRAIC[18]+ALGEBRAIC[31]);
RATES[8] = (ALGEBRAIC[5] - STATES[8])/ALGEBRAIC[40];
ALGEBRAIC[6] = 1.00000/pow(1.00000+exp((STATES[0]+71.5500)/7.43000), 2.00000);
ALGEBRAIC[19] = (STATES[0]<- 40.0000 ? (( ( - 25428.0*exp( 0.244400*STATES[0]) -  6.94800e-06*exp( - 0.0439100*STATES[0]))*(STATES[0]+37.7800))/1.00000)/(1.00000+exp( 0.311000*(STATES[0]+79.2300))) : 0.00000);
ALGEBRAIC[32] = (STATES[0]<- 40.0000 ? ( 0.0242400*exp( - 0.0105200*STATES[0]))/(1.00000+exp( - 0.137800*(STATES[0]+40.1400))) : ( 0.600000*exp( 0.0570000*STATES[0]))/(1.00000+exp( - 0.100000*(STATES[0]+32.0000))));
ALGEBRAIC[41] = 1.00000/(ALGEBRAIC[19]+ALGEBRAIC[32]);
RATES[9] = (ALGEBRAIC[6] - STATES[9])/ALGEBRAIC[41];
ALGEBRAIC[7] = 1.00000/(1.00000+exp((- 5.00000 - STATES[0])/7.50000));
ALGEBRAIC[20] = 1.40000/(1.00000+exp((- 35.0000 - STATES[0])/13.0000))+0.250000;
ALGEBRAIC[33] = 1.40000/(1.00000+exp((STATES[0]+5.00000)/5.00000));
ALGEBRAIC[42] = 1.00000/(1.00000+exp((50.0000 - STATES[0])/20.0000));
ALGEBRAIC[45] =  1.00000*ALGEBRAIC[20]*ALGEBRAIC[33]+ALGEBRAIC[42];
RATES[10] = (ALGEBRAIC[7] - STATES[10])/ALGEBRAIC[45];
ALGEBRAIC[9] = 1.00000/(1.00000+pow(STATES[3]/0.000325000, 8.00000));
ALGEBRAIC[22] = 0.100000/(1.00000+exp((STATES[3] - 0.000500000)/0.000100000));
ALGEBRAIC[34] = 0.200000/(1.00000+exp((STATES[3] - 0.000750000)/0.000800000));
ALGEBRAIC[43] = (ALGEBRAIC[9]+ALGEBRAIC[22]+ALGEBRAIC[34]+0.230000)/1.46000;
ALGEBRAIC[46] = (ALGEBRAIC[43] - STATES[12])/CONSTANTS[45];
RATES[12] = (ALGEBRAIC[43]>STATES[12]&&STATES[0]>- 60.0000 ? 0.00000 : ALGEBRAIC[46]);
ALGEBRAIC[58] = (( (( CONSTANTS[21]*CONSTANTS[10])/(CONSTANTS[10]+CONSTANTS[22]))*STATES[2])/(STATES[2]+CONSTANTS[23]))/(1.00000+ 0.124500*exp(( - 0.100000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))+ 0.0353000*exp(( - STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])));
ALGEBRAIC[13] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[11]/STATES[2]);
ALGEBRAIC[53] =  CONSTANTS[16]*pow(STATES[7], 3.00000)*STATES[8]*STATES[9]*(STATES[0] - ALGEBRAIC[13]);
ALGEBRAIC[54] =  CONSTANTS[17]*(STATES[0] - ALGEBRAIC[13]);
ALGEBRAIC[59] = ( CONSTANTS[24]*( exp(( CONSTANTS[27]*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))*pow(STATES[2], 3.00000)*CONSTANTS[12] -  exp(( (CONSTANTS[27] - 1.00000)*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))*pow(CONSTANTS[11], 3.00000)*STATES[3]*CONSTANTS[26]))/( (pow(CONSTANTS[29], 3.00000)+pow(CONSTANTS[11], 3.00000))*(CONSTANTS[28]+CONSTANTS[12])*(1.00000+ CONSTANTS[25]*exp(( (CONSTANTS[27] - 1.00000)*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))));
RATES[2] = ( - (ALGEBRAIC[53]+ALGEBRAIC[54]+ 3.00000*ALGEBRAIC[58]+ 3.00000*ALGEBRAIC[59])*CONSTANTS[3])/( CONSTANTS[4]*CONSTANTS[2]);
ALGEBRAIC[26] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[10]/STATES[1]);
ALGEBRAIC[47] = 0.100000/(1.00000+exp( 0.0600000*((STATES[0] - ALGEBRAIC[26]) - 200.000)));
ALGEBRAIC[48] = ( 3.00000*exp( 0.000200000*((STATES[0] - ALGEBRAIC[26])+100.000))+ 1.00000*exp( 0.100000*((STATES[0] - ALGEBRAIC[26]) - 10.0000)))/(1.00000+exp( - 0.500000*(STATES[0] - ALGEBRAIC[26])));
ALGEBRAIC[49] = ALGEBRAIC[47]/(ALGEBRAIC[47]+ALGEBRAIC[48]);
ALGEBRAIC[50] =  CONSTANTS[13]*ALGEBRAIC[49]* pow((CONSTANTS[10]/5.40000), 1.0 / 2)*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[57] =  CONSTANTS[20]*STATES[14]*STATES[13]*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[51] =  CONSTANTS[14]* pow((CONSTANTS[10]/5.40000), 1.0 / 2)*STATES[4]*STATES[5]*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[35] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log((CONSTANTS[10]+ CONSTANTS[9]*CONSTANTS[11])/(STATES[1]+ CONSTANTS[9]*STATES[2]));
ALGEBRAIC[52] =  CONSTANTS[15]*pow(STATES[6], 2.00000)*(STATES[0] - ALGEBRAIC[35]);
ALGEBRAIC[55] = ( (( CONSTANTS[18]*STATES[10]*STATES[11]*STATES[12]*4.00000*STATES[0]*pow(CONSTANTS[2], 2.00000))/( CONSTANTS[0]*CONSTANTS[1]))*( STATES[3]*exp(( 2.00000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])) -  0.341000*CONSTANTS[12]))/(exp(( 2.00000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])) - 1.00000);
ALGEBRAIC[44] =  (( 0.500000*CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[12]/STATES[3]);
ALGEBRAIC[56] =  CONSTANTS[19]*(STATES[0] - ALGEBRAIC[44]);
ALGEBRAIC[61] = ( CONSTANTS[32]*(STATES[0] - ALGEBRAIC[26]))/(1.00000+exp((25.0000 - STATES[0])/5.98000));
ALGEBRAIC[60] = ( CONSTANTS[30]*STATES[3])/(STATES[3]+CONSTANTS[31]);
ALGEBRAIC[0] = (VOI -  floor(VOI/CONSTANTS[6])*CONSTANTS[6]>=CONSTANTS[5]&&VOI -  floor(VOI/CONSTANTS[6])*CONSTANTS[6]<=CONSTANTS[5]+CONSTANTS[7] ? CONSTANTS[8] : 0.00000);
RATES[0] = - (ALGEBRAIC[50]+ALGEBRAIC[57]+ALGEBRAIC[51]+ALGEBRAIC[52]+ALGEBRAIC[55]+ALGEBRAIC[58]+ALGEBRAIC[53]+ALGEBRAIC[54]+ALGEBRAIC[59]+ALGEBRAIC[56]+ALGEBRAIC[61]+ALGEBRAIC[60]+ALGEBRAIC[0]);
RATES[1] = ( - ((ALGEBRAIC[50]+ALGEBRAIC[57]+ALGEBRAIC[51]+ALGEBRAIC[52]+ALGEBRAIC[61]+ALGEBRAIC[0]) -  2.00000*ALGEBRAIC[58])*CONSTANTS[3])/( CONSTANTS[4]*CONSTANTS[2]);
ALGEBRAIC[62] =  (( CONSTANTS[34]*pow(STATES[15], 2.00000))/(pow(CONSTANTS[35], 2.00000)+pow(STATES[15], 2.00000))+CONSTANTS[36])*STATES[10]*STATES[16];
ALGEBRAIC[63] = CONSTANTS[39]/(1.00000+pow(CONSTANTS[37], 2.00000)/pow(STATES[3], 2.00000));
ALGEBRAIC[64] =  CONSTANTS[38]*(STATES[15] - STATES[3]);
ALGEBRAIC[65] = (( (- ((ALGEBRAIC[55]+ALGEBRAIC[56]+ALGEBRAIC[60]) -  2.00000*ALGEBRAIC[59])/( 2.00000*CONSTANTS[4]*CONSTANTS[2]))*CONSTANTS[3]+ALGEBRAIC[64]) - ALGEBRAIC[63])+ALGEBRAIC[62];
ALGEBRAIC[67] = 1.00000/(1.00000+( CONSTANTS[40]*CONSTANTS[41])/pow(STATES[3]+CONSTANTS[41], 2.00000));
RATES[3] =  ALGEBRAIC[65]*ALGEBRAIC[67];
ALGEBRAIC[66] =  (CONSTANTS[4]/CONSTANTS[44])*(ALGEBRAIC[63] - (ALGEBRAIC[62]+ALGEBRAIC[64]));
ALGEBRAIC[68] = 1.00000/(1.00000+( CONSTANTS[42]*CONSTANTS[43])/pow(STATES[15]+CONSTANTS[43], 2.00000));
RATES[15] =  ALGEBRAIC[66]*ALGEBRAIC[68];
}
__global__ void computeVariables(double VOI, double* CONSTANTS, double* RATES, double* STATES, double* ALGEBRAIC)
{
ALGEBRAIC[8] = 1.00000/(1.00000+exp((STATES[0]+20.0000)/7.00000));
ALGEBRAIC[21] =  1125.00*exp(- pow(STATES[0]+27.0000, 2.00000)/240.000)+80.0000+165.000/(1.00000+exp((25.0000 - STATES[0])/10.0000));
ALGEBRAIC[10] = 1.00000/(1.00000+exp((STATES[0]+20.0000)/5.00000));
ALGEBRAIC[23] =  85.0000*exp(- pow(STATES[0]+45.0000, 2.00000)/320.000)+5.00000/(1.00000+exp((STATES[0] - 20.0000)/5.00000))+3.00000;
ALGEBRAIC[11] = 1.00000/(1.00000+exp((20.0000 - STATES[0])/6.00000));
ALGEBRAIC[24] =  9.50000*exp(- pow(STATES[0]+40.0000, 2.00000)/1800.00)+0.800000;
ALGEBRAIC[12] = (STATES[3]<0.000350000 ? 1.00000/(1.00000+pow(STATES[3]/0.000350000, 6.00000)) : 1.00000/(1.00000+pow(STATES[3]/0.000350000, 16.0000)));
ALGEBRAIC[25] = (ALGEBRAIC[12] - STATES[16])/CONSTANTS[33];
ALGEBRAIC[1] = 1.00000/(1.00000+exp((- 26.0000 - STATES[0])/7.00000));
ALGEBRAIC[14] = 450.000/(1.00000+exp((- 45.0000 - STATES[0])/10.0000));
ALGEBRAIC[27] = 6.00000/(1.00000+exp((STATES[0]+30.0000)/11.5000));
ALGEBRAIC[36] =  1.00000*ALGEBRAIC[14]*ALGEBRAIC[27];
ALGEBRAIC[2] = 1.00000/(1.00000+exp((STATES[0]+88.0000)/24.0000));
ALGEBRAIC[15] = 3.00000/(1.00000+exp((- 60.0000 - STATES[0])/20.0000));
ALGEBRAIC[28] = 1.12000/(1.00000+exp((STATES[0] - 60.0000)/20.0000));
ALGEBRAIC[37] =  1.00000*ALGEBRAIC[15]*ALGEBRAIC[28];
ALGEBRAIC[3] = 1.00000/(1.00000+exp((- 5.00000 - STATES[0])/14.0000));
ALGEBRAIC[16] = 1100.00/ pow((1.00000+exp((- 10.0000 - STATES[0])/6.00000)), 1.0 / 2);
ALGEBRAIC[29] = 1.00000/(1.00000+exp((STATES[0] - 60.0000)/20.0000));
ALGEBRAIC[38] =  1.00000*ALGEBRAIC[16]*ALGEBRAIC[29];
ALGEBRAIC[4] = 1.00000/pow(1.00000+exp((- 56.8600 - STATES[0])/9.03000), 2.00000);
ALGEBRAIC[17] = 1.00000/(1.00000+exp((- 60.0000 - STATES[0])/5.00000));
ALGEBRAIC[30] = 0.100000/(1.00000+exp((STATES[0]+35.0000)/5.00000))+0.100000/(1.00000+exp((STATES[0] - 50.0000)/200.000));
ALGEBRAIC[39] =  1.00000*ALGEBRAIC[17]*ALGEBRAIC[30];
ALGEBRAIC[5] = 1.00000/pow(1.00000+exp((STATES[0]+71.5500)/7.43000), 2.00000);
ALGEBRAIC[18] = (STATES[0]<- 40.0000 ?  0.0570000*exp(- (STATES[0]+80.0000)/6.80000) : 0.00000);
ALGEBRAIC[31] = (STATES[0]<- 40.0000 ?  2.70000*exp( 0.0790000*STATES[0])+ 310000.*exp( 0.348500*STATES[0]) : 0.770000/( 0.130000*(1.00000+exp((STATES[0]+10.6600)/- 11.1000))));
ALGEBRAIC[40] = 1.00000/(ALGEBRAIC[18]+ALGEBRAIC[31]);
ALGEBRAIC[6] = 1.00000/pow(1.00000+exp((STATES[0]+71.5500)/7.43000), 2.00000);
ALGEBRAIC[19] = (STATES[0]<- 40.0000 ? (( ( - 25428.0*exp( 0.244400*STATES[0]) -  6.94800e-06*exp( - 0.0439100*STATES[0]))*(STATES[0]+37.7800))/1.00000)/(1.00000+exp( 0.311000*(STATES[0]+79.2300))) : 0.00000);
ALGEBRAIC[32] = (STATES[0]<- 40.0000 ? ( 0.0242400*exp( - 0.0105200*STATES[0]))/(1.00000+exp( - 0.137800*(STATES[0]+40.1400))) : ( 0.600000*exp( 0.0570000*STATES[0]))/(1.00000+exp( - 0.100000*(STATES[0]+32.0000))));
ALGEBRAIC[41] = 1.00000/(ALGEBRAIC[19]+ALGEBRAIC[32]);
ALGEBRAIC[7] = 1.00000/(1.00000+exp((- 5.00000 - STATES[0])/7.50000));
ALGEBRAIC[20] = 1.40000/(1.00000+exp((- 35.0000 - STATES[0])/13.0000))+0.250000;
ALGEBRAIC[33] = 1.40000/(1.00000+exp((STATES[0]+5.00000)/5.00000));
ALGEBRAIC[42] = 1.00000/(1.00000+exp((50.0000 - STATES[0])/20.0000));
ALGEBRAIC[45] =  1.00000*ALGEBRAIC[20]*ALGEBRAIC[33]+ALGEBRAIC[42];
ALGEBRAIC[9] = 1.00000/(1.00000+pow(STATES[3]/0.000325000, 8.00000));
ALGEBRAIC[22] = 0.100000/(1.00000+exp((STATES[3] - 0.000500000)/0.000100000));
ALGEBRAIC[34] = 0.200000/(1.00000+exp((STATES[3] - 0.000750000)/0.000800000));
ALGEBRAIC[43] = (ALGEBRAIC[9]+ALGEBRAIC[22]+ALGEBRAIC[34]+0.230000)/1.46000;
ALGEBRAIC[46] = (ALGEBRAIC[43] - STATES[12])/CONSTANTS[45];
ALGEBRAIC[58] = (( (( CONSTANTS[21]*CONSTANTS[10])/(CONSTANTS[10]+CONSTANTS[22]))*STATES[2])/(STATES[2]+CONSTANTS[23]))/(1.00000+ 0.124500*exp(( - 0.100000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))+ 0.0353000*exp(( - STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])));
ALGEBRAIC[13] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[11]/STATES[2]);
ALGEBRAIC[53] =  CONSTANTS[16]*pow(STATES[7], 3.00000)*STATES[8]*STATES[9]*(STATES[0] - ALGEBRAIC[13]);
ALGEBRAIC[54] =  CONSTANTS[17]*(STATES[0] - ALGEBRAIC[13]);
ALGEBRAIC[59] = ( CONSTANTS[24]*( exp(( CONSTANTS[27]*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))*pow(STATES[2], 3.00000)*CONSTANTS[12] -  exp(( (CONSTANTS[27] - 1.00000)*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))*pow(CONSTANTS[11], 3.00000)*STATES[3]*CONSTANTS[26]))/( (pow(CONSTANTS[29], 3.00000)+pow(CONSTANTS[11], 3.00000))*(CONSTANTS[28]+CONSTANTS[12])*(1.00000+ CONSTANTS[25]*exp(( (CONSTANTS[27] - 1.00000)*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1]))));
ALGEBRAIC[26] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[10]/STATES[1]);
ALGEBRAIC[47] = 0.100000/(1.00000+exp( 0.0600000*((STATES[0] - ALGEBRAIC[26]) - 200.000)));
ALGEBRAIC[48] = ( 3.00000*exp( 0.000200000*((STATES[0] - ALGEBRAIC[26])+100.000))+ 1.00000*exp( 0.100000*((STATES[0] - ALGEBRAIC[26]) - 10.0000)))/(1.00000+exp( - 0.500000*(STATES[0] - ALGEBRAIC[26])));
ALGEBRAIC[49] = ALGEBRAIC[47]/(ALGEBRAIC[47]+ALGEBRAIC[48]);
ALGEBRAIC[50] =  CONSTANTS[13]*ALGEBRAIC[49]* pow((CONSTANTS[10]/5.40000), 1.0 / 2)*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[57] =  CONSTANTS[20]*STATES[14]*STATES[13]*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[51] =  CONSTANTS[14]* pow((CONSTANTS[10]/5.40000), 1.0 / 2)*STATES[4]*STATES[5]*(STATES[0] - ALGEBRAIC[26]);
ALGEBRAIC[35] =  (( CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log((CONSTANTS[10]+ CONSTANTS[9]*CONSTANTS[11])/(STATES[1]+ CONSTANTS[9]*STATES[2]));
ALGEBRAIC[52] =  CONSTANTS[15]*pow(STATES[6], 2.00000)*(STATES[0] - ALGEBRAIC[35]);
ALGEBRAIC[55] = ( (( CONSTANTS[18]*STATES[10]*STATES[11]*STATES[12]*4.00000*STATES[0]*pow(CONSTANTS[2], 2.00000))/( CONSTANTS[0]*CONSTANTS[1]))*( STATES[3]*exp(( 2.00000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])) -  0.341000*CONSTANTS[12]))/(exp(( 2.00000*STATES[0]*CONSTANTS[2])/( CONSTANTS[0]*CONSTANTS[1])) - 1.00000);
ALGEBRAIC[44] =  (( 0.500000*CONSTANTS[0]*CONSTANTS[1])/CONSTANTS[2])*log(CONSTANTS[12]/STATES[3]);
ALGEBRAIC[56] =  CONSTANTS[19]*(STATES[0] - ALGEBRAIC[44]);
ALGEBRAIC[61] = ( CONSTANTS[32]*(STATES[0] - ALGEBRAIC[26]))/(1.00000+exp((25.0000 - STATES[0])/5.98000));
ALGEBRAIC[60] = ( CONSTANTS[30]*STATES[3])/(STATES[3]+CONSTANTS[31]);
ALGEBRAIC[0] = (VOI -  floor(VOI/CONSTANTS[6])*CONSTANTS[6]>=CONSTANTS[5]&&VOI -  floor(VOI/CONSTANTS[6])*CONSTANTS[6]<=CONSTANTS[5]+CONSTANTS[7] ? CONSTANTS[8] : 0.00000);
ALGEBRAIC[62] =  (( CONSTANTS[34]*pow(STATES[15], 2.00000))/(pow(CONSTANTS[35], 2.00000)+pow(STATES[15], 2.00000))+CONSTANTS[36])*STATES[10]*STATES[16];
ALGEBRAIC[63] = CONSTANTS[39]/(1.00000+pow(CONSTANTS[37], 2.00000)/pow(STATES[3], 2.00000));
ALGEBRAIC[64] =  CONSTANTS[38]*(STATES[15] - STATES[3]);
ALGEBRAIC[65] = (( (- ((ALGEBRAIC[55]+ALGEBRAIC[56]+ALGEBRAIC[60]) -  2.00000*ALGEBRAIC[59])/( 2.00000*CONSTANTS[4]*CONSTANTS[2]))*CONSTANTS[3]+ALGEBRAIC[64]) - ALGEBRAIC[63])+ALGEBRAIC[62];
ALGEBRAIC[67] = 1.00000/(1.00000+( CONSTANTS[40]*CONSTANTS[41])/pow(STATES[3]+CONSTANTS[41], 2.00000));
ALGEBRAIC[66] =  (CONSTANTS[4]/CONSTANTS[44])*(ALGEBRAIC[63] - (ALGEBRAIC[62]+ALGEBRAIC[64]));
ALGEBRAIC[68] = 1.00000/(1.00000+( CONSTANTS[42]*CONSTANTS[43])/pow(STATES[15]+CONSTANTS[43], 2.00000));
}

drug_t ic50;
__device__ drug_t *d_ic50;
// double ic50[2000][14];
// double *d_ic50[2000][14];

double *d_concs[4];
__device__ double *d_time_step;

// __global__ void toc(clock_t start = START_TIMER);

__global__ void check_data(){
  printf("check data: \n");
  int idx = 14;
  for(int sample_index=0; sample_index<idx; sample_index++){
        printf("%lf|", d_ic50[2][sample_index]);
        }
     //   printf("\n \n");
}

__global__ void set_time_step(
  /*
  as 'adaptive' solver, we need the time step to change in the middle of 
  the process
  since we need to change almost every function to void, I change the 
  return time_step to 
  cudaMemCopy the time_step, 
  */
    double TIME,
    double time_point,
    double max_time_step,
    double* CONSTANTS,
    double* RATES,
    double* STATES,
    double* ALGEBRAIC) {
    double time_step = 0.005;

    if (TIME <= time_point || (TIME - floor(TIME / CONSTANTS[stim_period]) * CONSTANTS[stim_period]) <= time_point) {
        //printf("TIME <= time_point ms\n");
        //return time_step;
        memcpy(d_time_step, &time_step, sizeof(double));
        __syncthreads(); //equivalent to break
        //printf("dV = %lf, time_step = %lf\n",RATES[V] * time_step, time_step);
    }
    else {
        //printf("TIME > time_point ms\n");
        if (std::abs(RATES[V] * time_step) <= 0.2) {//Slow changes in V
            //printf("dV/dt <= 0.2\n");
            time_step = std::abs(0.8 / RATES[V]);
            //Make sure time_step is between 0.005 and max_time_step
            if (time_step < 0.005) {
                time_step = 0.005;
            }
            else if (time_step > max_time_step) {
                time_step = max_time_step;
            }
            //printf("dV = %lf, time_step = %lf\n",std::abs(RATES[V] * time_step), time_step);
        }
        else if (std::abs(RATES[V] * time_step) >= 0.8) {//Fast changes in V
            //printf("dV/dt >= 0.8\n");
            time_step = std::abs(0.2 / RATES[V]);
            while (std::abs(RATES[V] * time_step) >= 0.8 && 0.005 < time_step && time_step < max_time_step) {
                time_step = time_step / 10.0;
                //printf("dV = %lf, time_step = %lf\n",std::abs(RATES[V] * time_step), time_step);
            }
        }
        // return time_step;
        memcpy(d_time_step, &time_step, sizeof(double));
    }
}
__global__ void solveAnalytical(double* CONSTANTS, double* RATES, double *STATES, double *ALGEBRAIC, double dt)
{
  ////==============
  ////Exact solution
  ////==============
  ////INa
  STATES[m] = ALGEBRAIC[mss] - (ALGEBRAIC[mss] - STATES[m]) * exp(-dt / ALGEBRAIC[tm]);
  STATES[hf] = ALGEBRAIC[hss] - (ALGEBRAIC[hss] - STATES[hf]) * exp(-dt / ALGEBRAIC[thf]);
  STATES[hs] = ALGEBRAIC[hss] - (ALGEBRAIC[hss] - STATES[hs]) * exp(-dt / ALGEBRAIC[ths]);
  STATES[j] = ALGEBRAIC[jss] - (ALGEBRAIC[jss] - STATES[j]) * exp(-dt / ALGEBRAIC[tj]);
  STATES[hsp] = ALGEBRAIC[hssp] - (ALGEBRAIC[hssp] - STATES[hsp]) * exp(-dt / ALGEBRAIC[thsp]);
  STATES[jp] = ALGEBRAIC[jss] - (ALGEBRAIC[jss] - STATES[jp]) * exp(-dt / ALGEBRAIC[tjp]);
  STATES[mL] = ALGEBRAIC[mLss] - (ALGEBRAIC[mLss] - STATES[mL]) * exp(-dt / ALGEBRAIC[tmL]);
  STATES[hL] = ALGEBRAIC[hLss] - (ALGEBRAIC[hLss] - STATES[hL]) * exp(-dt / CONSTANTS[thL]);
  STATES[hLp] = ALGEBRAIC[hLssp] - (ALGEBRAIC[hLssp] - STATES[hLp]) * exp(-dt / CONSTANTS[thLp]);
  ////Ito
  STATES[a] = ALGEBRAIC[ass] - (ALGEBRAIC[ass] - STATES[a]) * exp(-dt / ALGEBRAIC[ta]);
  STATES[iF] = ALGEBRAIC[iss] - (ALGEBRAIC[iss] - STATES[iF]) * exp(-dt / ALGEBRAIC[tiF]);
  STATES[iS] = ALGEBRAIC[iss] - (ALGEBRAIC[iss] - STATES[iS]) * exp(-dt / ALGEBRAIC[tiS]);
  STATES[ap] = ALGEBRAIC[assp] - (ALGEBRAIC[assp] - STATES[ap]) * exp(-dt / ALGEBRAIC[ta]);
  STATES[iFp] = ALGEBRAIC[iss] - (ALGEBRAIC[iss] - STATES[iFp]) * exp(-dt / ALGEBRAIC[tiFp]);
  STATES[iSp] = ALGEBRAIC[iss] - (ALGEBRAIC[iss] - STATES[iSp]) * exp(-dt / ALGEBRAIC[tiSp]);
  ////ICaL
  STATES[d] = ALGEBRAIC[dss] - (ALGEBRAIC[dss] - STATES[d]) * exp(-dt / ALGEBRAIC[td]);
  STATES[ff] = ALGEBRAIC[fss] - (ALGEBRAIC[fss] - STATES[ff]) * exp(-dt / ALGEBRAIC[tff]);
  STATES[fs] = ALGEBRAIC[fss] - (ALGEBRAIC[fss] - STATES[fs]) * exp(-dt / ALGEBRAIC[tfs]);
  STATES[fcaf] = ALGEBRAIC[fcass] - (ALGEBRAIC[fcass] - STATES[fcaf]) * exp(-dt / ALGEBRAIC[tfcaf]);
  STATES[fcas] = ALGEBRAIC[fcass] - (ALGEBRAIC[fcass] - STATES[fcas]) * exp(-dt / ALGEBRAIC[tfcas]);
  STATES[jca] = ALGEBRAIC[fcass] - (ALGEBRAIC[fcass] - STATES[jca]) * exp(- dt / CONSTANTS[tjca]);
  STATES[ffp] = ALGEBRAIC[fss] - (ALGEBRAIC[fss] - STATES[ffp]) * exp(-dt / ALGEBRAIC[tffp]);
  STATES[fcafp] = ALGEBRAIC[fcass] - (ALGEBRAIC[fcass] - STATES[fcafp]) * exp(-d / ALGEBRAIC[tfcafp]);
  STATES[nca] = ALGEBRAIC[anca] * CONSTANTS[k2n] / ALGEBRAIC[km2n] -
      (ALGEBRAIC[anca] * CONSTANTS[k2n] / ALGEBRAIC[km2n] - STATES[nca]) * exp(-ALGEBRAIC[km2n] * dt);
  ////IKr
  STATES[xrf] = ALGEBRAIC[xrss] - (ALGEBRAIC[xrss] - STATES[xrf]) * exp(-dt / ALGEBRAIC[txrf]);
  STATES[xrs] = ALGEBRAIC[xrss] - (ALGEBRAIC[xrss] - STATES[xrs]) * exp(-dt / ALGEBRAIC[txrs]);
  ////IKs
  STATES[xs1] = ALGEBRAIC[xs1ss] - (ALGEBRAIC[xs1ss] - STATES[xs1]) * exp(-dt / ALGEBRAIC[txs1]);
  STATES[xs2] = ALGEBRAIC[xs2ss] - (ALGEBRAIC[xs2ss] - STATES[xs2]) * exp(-dt / ALGEBRAIC[txs2]);
  ////IK1
  STATES[xk1] = ALGEBRAIC[xk1ss] - (ALGEBRAIC[xk1ss] - STATES[xk1]) * exp(-dt / ALGEBRAIC[txk1]);
  ////INaCa
  ////INaK
  ////IKb
  ////INab
  ////ICab
  ///IpCa
  ////Diffusion fluxes
  ////RyR receptors
  STATES[Jrelnp] = ALGEBRAIC[Jrel_inf] - (ALGEBRAIC[Jrel_inf] - STATES[Jrelnp]) * exp(-dt / ALGEBRAIC[tau_rel]);
  STATES[Jrelp] = ALGEBRAIC[Jrel_infp] - (ALGEBRAIC[Jrel_infp] - STATES[Jrelp]) * exp(-dt / ALGEBRAIC[tau_relp]);
  ////SERCA Pump
  ////Calcium translocation
  //
  ////=============================
  ////Approximated solution (Euler)
  ////=============================
  ////ICaL
  //STATES[jca] = STATES[jca] + RATES[jca] * dt;
  ////CaMK
  STATES[CaMKt] = STATES[CaMKt] + RATES[CaMKt] * dt;
  ////Membrane potential
  STATES[V] = STATES[V] + RATES[V] * dt;
  ////Ion Concentrations and Buffers
  STATES[nai] = STATES[nai] + RATES[nai] * dt;
  STATES[nass] = STATES[nass] + RATES[nass] * dt;
  STATES[ki] = STATES[ki] + RATES[ki] * dt;
  STATES[kss] = STATES[kss] + RATES[kss] * dt;
  STATES[cai] = STATES[cai] + RATES[cai] * dt;
  STATES[cass] = STATES[cass] + RATES[cass] * dt;
  STATES[cansr] = STATES[cansr] + RATES[cansr] * dt;
  STATES[cajsr] = STATES[cajsr] + RATES[cajsr] * dt; 
  //========================
  //Full Euler Approximation
  //========================
  //STATES[V] = STATES[V] + RATES[V] * dt;
  //STATES[CaMKt] = STATES[CaMKt] + RATES[CaMKt] * dt;
  //STATES[cass] = STATES[cass] + RATES[cass] * dt;
  //STATES[nai] = STATES[nai] + RATES[nai] * dt;
  //STATES[nass] = STATES[nass] + RATES[nass] * dt;
  //STATES[ki] = STATES[ki] + RATES[ki] * dt;
  //STATES[kss] = STATES[kss] + RATES[kss] * dt;
  //STATES[cansr] = STATES[cansr] + RATES[cansr] * dt;
  //STATES[cajsr] = STATES[cajsr] + RATES[cajsr] * dt;
  //STATES[cai] = STATES[cai] + RATES[cai] * dt;
  //STATES[m] = STATES[m] + RATES[m] * dt;
  //STATES[hf] = STATES[hf] + RATES[hf] * dt;
  //STATES[hs] = STATES[hs] + RATES[hs] * dt;
  //STATES[j] = STATES[j] + RATES[j] * dt;
  //STATES[hsp] = STATES[hsp] + RATES[hsp] * dt;
  //STATES[jp] = STATES[jp] + RATES[jp] * dt;
  //STATES[mL] = STATES[mL] + RATES[mL] * dt;
  //STATES[hL] = STATES[hL] + RATES[hL] * dt;
  //STATES[hLp] = STATES[hLp] + RATES[hLp] * dt;
  //STATES[a] = STATES[a] + RATES[a] * dt;
  //STATES[iF] = STATES[iF] + RATES[iF] * dt;
  //STATES[iS] = STATES[iS] + RATES[iS] * dt;
  //STATES[ap] = STATES[ap] + RATES[ap] * dt;
  //STATES[iFp] = STATES[iFp] + RATES[iFp] * dt;
  //STATES[iSp] = STATES[iSp] + RATES[iSp] * dt;
  //STATES[d] = STATES[d] + RATES[d] * dt;
  //STATES[ff] = STATES[ff] + RATES[ff] * dt;
  //STATES[fs] = STATES[fs] + RATES[fs] * dt;
  //STATES[fcaf] = STATES[fcaf] + RATES[fcaf] * dt;
  //STATES[fcas] = STATES[fcas] + RATES[fcas] * dt;
  //STATES[jca] = STATES[jca] + RATES[jca] * dt;
  //STATES[ffp] = STATES[ffp] + RATES[ffp] * dt;
  //STATES[fcafp] = STATES[fcafp] + RATES[fcafp] * dt;
  //STATES[nca] = STATES[nca] + RATES[nca] * dt;
  //STATES[xrf] = STATES[xrf] + RATES[xrf] * dt;
  //STATES[xrs] = STATES[xrs] + RATES[xrs] * dt;
  //STATES[xs1] = STATES[xs1] + RATES[xs1] * dt;
  //STATES[xs2] = STATES[xs2] + RATES[xs2] * dt;
  //STATES[xk1] = STATES[xk1] + RATES[xk1] * dt;
  //STATES[Jrelnp] = STATES[Jrelnp] + RATES[Jrelnp] * dt;
  //STATES[Jrelp] = STATES[Jrelp] + RATES[Jrelp] * dt;
}



__global__ void do_drug_sim_analytical(double conc,const param_t* p_param, 
const unsigned short sample_id)
{

  /*
  do drug effect simulation, loop will be replaced with kernel loops
  */
  double tcurr = 0.0, dt = 0.005, dt_set, tmax;
  double max_time_step = 1.0, time_point = 25.0;

  double ic50[14] = {2704.000000, 0.695400, 0.000000, 0.000000,
          50490.000000, 0.627700, 2371.000000, 1.984000,
          1947.000000, 1.473000, 12460.000000, 2.885000,
          53.100000, 1.075000};
  
  // files for storing results
  // time-series result
  FILE *vfp_m, *fp_inet, *fp_gate;

  // features
  double inet, qnet;

  // looping counter
  unsigned short idx = 14;
  
  // simulation parameters
  double dtw = 2.0;
  const char *drug_name = "bepridil";
  const double bcl = 2000;
  const double inet_vm_threshold = -88.0;
  const unsigned short pace_max = 10;
  const unsigned short celltype = 0.;
  const unsigned short last_pace_print = 3;
  const unsigned short last_drug_check_pace = 250;
  const unsigned int print_freq = (1./dt) * dtw;
  unsigned short pace_count = 0;
  unsigned short pace_steepest = 0;
  double* RATES;
  double* STATES;
  double* CONSTANTS;
  double* ALGEBRAIC;
  int num_of_algebraic = 69;
  int num_of_constants = 46;
  int num_of_rates = 17;
  int num_of_states = 17;

  RATES = (double*)malloc((num_of_rates)*sizeof(double));
  STATES = (double*)malloc((num_of_states)*sizeof(double));
  CONSTANTS = (double*)malloc((num_of_constants)*sizeof(double));
  ALGEBRAIC = (double*)malloc((num_of_algebraic)*sizeof(double));

  // apply some cell initialization
  initConsts<<<1,1>>>(CONSTANTS, RATES, STATES);
  //p_cell->initConsts( celltype, conc, ic50.data());
  CONSTANTS[stim_period] = bcl;

  // generate file for time-series output
  // snprintf(buffer, sizeof(buffer), "result/%s_%.2lf_vmcheck_smp%d.plt", 
  //           drug_name, conc, sample_id );
  // fp_vm = fopen( buffer, "w" );
  // snprintf(buffer, sizeof(buffer), "result/%s_%.2lf_gates_smp%d.plt",
  //           drug_name, conc, sample_id);
  // fp_gate = fopen(buffer, "w");
  // printf("drug name: %s , concentration: %.2lf , sample id: %d \n", drug_name, conc, sample_id);
  printf("\n");

  // printf(fp_vm, "%s %s\n", "Time", "Vm");
  //printf("Time: %s Vm: %s\n", "Time", "Vm");
  // fprintf(fp_gate, "Time %s\n", GATES_HEADER); //this is to write headers in results

  tmax = pace_max * bcl;

  while (tcurr < tmax) {
    // dt_set = set_time_step<<<1,1>>>(tcurr,
    //     		   time_point,
		//            max_time_step,
  	// 	         CONSTANTS,
		//            RATES,
		// 	         STATES,
		//            ALGEBRAIC);
    set_time_step<<<1,1>>>(tcurr,
        		   time_point,
		           max_time_step,
  		         CONSTANTS,
		           RATES,
			         STATES,
		           ALGEBRAIC);
              // hipDeviceSynchronize();
    // printf("set time step\n");
    //printf("timestep pointer: %x \n",d_time_step);
    //dt_set = *d_time_step;
    dt_set = 0.0001;

    // // //Compute all rates at tcurr
    computeRates<<<1,1>>>(tcurr,
		          CONSTANTS,
            	RATES,
		          STATES,
            	ALGEBRAIC);
              // hipDeviceSynchronize();
    // printf("compute rates at tcurr\n");

    //Compute the correct/accepted time step
    if (floor((tcurr + dt_set) / bcl) == floor(tcurr / bcl)) {
      dt = dt_set;
    }
    else {
      dt = (floor(tcurr / bcl) + 1) * bcl - tcurr;
    }

    //Compute the analytical solution
    solveAnalytical<<<1,1>>>(CONSTANTS, RATES, STATES, ALGEBRAIC, dt);
    //printf("solve analytical done\n");
    
    //=============//
    //Print results//
    //=============//
    // fprintf(fp_vm, "%lf %lf\n", tcurr, STATES[V]);
    // fprintf(fp_gate, "%lf ",tcurr);
    printf("tcurr: %lf States[v]: %lf\n", tcurr, STATES[V]);
    // printf("%lf \n \n",tcurr);    
    // for(idx = 0; idx < p_cell->gates_size; idx++){
    //   fprintf(fp_gate, "%lf ", p_cell->STATES[p_cell->GATES_INDICES[idx]]);
    // }
    // fprintf(fp_gate, "\n");
    printf("\n");
    
    //Next time step
    tcurr = tcurr + dt;
  }

  // clean the memories
  //fclose(fp_vm);
  //fclose(fp_gate);
}


//__global__ void Calculate(double d_ic50[11][14], double concs[4], Cellmodel *p_cell);
__global__ void Concentration(drug_t *d_ic50, double *concs[4]){
  
  /*
  uses block and thread in CUDA to replace concentration loop
  */

  // Get the thread ID.
  int sample_id = threadIdx.x;
  int conc_idx = blockIdx.x;
  //printf("doing calculation loop....\n");
  

  //for now, we hard code the concs
  double h_concs[4] = {0.0, 33.0, 66.0, 99.0};

  //memset(h_concs, -1, sizeof(h_concs));
  //printf("%lf", h_concs[1]);
  // hipMemcpy(d_p_cell, p_cell, sizeof(Cellmodel), hipMemcpyHostToDevice);
  // hipMemcpy(h_concs, concs, 4*sizeof(double), hipMemcpyDeviceToHost);

  // printf("concentration: %d -> value: %lf\n",conc_idx, h_concs[conc_idx]);
  // printf("Sample_ID: %d\n",sample_id );
  
  
  //       printf("\n");
        // for( const auto &conc: concs )
        // { // begin concentration loop
        // printf("Current Concentration: %lf  ",concs[a]);
        // // execute main simulation function
        // //do_drug_sim(conc, ic50[sample_id],
        // //            NULL, sample_id,
        // //            p_cell, ode_solver, cvode_firsttime);
        // // TODO @IritaSee: paralelise this loop that takes each data 
        
        //WARNING: concs still hard coded
       //do_drug_sim_analytical<<<1,1>>>(h_concs[conc_idx], *d_ic50[sample_id], NULL, sample_id);
       do_drug_sim_analytical<<<1,1>>>(h_concs[conc_idx], NULL, sample_id);

        // } // end concentration loop

}


int main()
{

    // input variables for cell simulation
    double bcl, dt;
    unsigned short pace;

    //prepare memory slots for ic_50 
    hipSetDevice(0);
    hipMalloc((drug_t**)&d_ic50, sizeof(drug_t));
    //perpare memory slots for concentration and copy it to the just created mem slots
    hipMalloc((void**)&d_concs, 4*sizeof(double)); 
    hipMemcpy(d_concs, concs, 4*sizeof(double), hipMemcpyHostToDevice);
    //prepare memory slots for p_cell and copy it
    // hipMalloc((void**)d_p_cell, sizeof(Cellmodel));
    // hipMemcpy(d_p_cell, p_cell, sizeof(Cellmodel), hipMemcpyHostToDevice);
    unsigned short idx;
    tic();
    snprintf(buffer, sizeof(buffer),
      "./drugs/bepridil/IC50_samples10.csv");
    //drug_t ic50 = get_IC50_data_from_file(buffer);
    //int data_row = sizeof(ic50)/sizeof(ic50[0]);
    int data_row = 10;
    get_IC50_data_from_file(buffer);
    if(sizeof(ic50)/sizeof(ic50[0]) == 0)
        printf("Something problem with the IC50 file!\n");
    else if(sizeof(ic50)/sizeof(ic50[0]) > 2000)
        printf("Too much input! Maximum sample data is 2000!\n");
    printf("start calculation....\n");
    // dim3 block(32,32);
    //dim3 grid ((columns+block.x-1)/block.x,(rows+block.y-1)/block.y);
    Concentration<<<4,data_row>>>(d_ic50, d_concs );  
    // Calculate(d_ic50, d_concs, d_p_cell );
    //concentration loop fails so i loop it altogether
    hipDeviceSynchronize();
    toc(START_TIMER);
    // loop to do calculation in each data is replaced by this func
    
    // memory cleaning and finalize the program
    

    return 0;
}

void get_IC50_data_from_file(const char* file_name)
{
  /*get IC50 data from a file*/
  /*caution: keep it host function!*/
  FILE *fp_drugs;
  printf("Reading the data....\n");
  
  char *token;
  //std::array<double,14> temp_array; //make the d_ version as well?
  double temp_array[1][14];
  //unsigned short idx;
  unsigned int idx;

  if( (fp_drugs = fopen(file_name, "r")) == NULL){
    printf("Cannot open file %s\n",
      file_name);
    //return ic50;
  }

  int count = 0;

  fgets(buffer, sizeof(buffer), fp_drugs); // skip header
  while( fgets(buffer, sizeof(buffer), fp_drugs) != NULL )
  { // begin line reading
    token = strtok( buffer, "," );
    idx = 0;
    while( token != NULL )
    { // begin data tokenizing
      temp_array[0][idx] = strtod(token, NULL);
      token = strtok(NULL, ",");
      ic50[count][idx] = temp_array[0][idx];
      idx=idx+1;
    } // end data tokenizing
    for(int sample_index=0; sample_index<idx; sample_index++){
        printf("%lf|", ic50[count][sample_index]);
        }
        printf("\n \n");
    //ic50.push_back(temp_array);
    count = count+1;
  } // end line reading

  fclose(fp_drugs);

  //copy the ic50 to GPU memory
  printf("rows found: %d\n",idx);
  
  hipMemcpy(d_ic50, ic50, idx * sizeof(drug_t), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  check_data<<<1,1>>>();
  //  printf("device memory sample contents: ");
  //  for(int sample_index=0; sample_index<idx; sample_index++){
  //       printf("%lf|", *d_ic50[1][sample_index]);
  //       }
  //       printf("\n \n");

  //return ic50;
}
