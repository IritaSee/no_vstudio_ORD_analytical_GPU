#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;
// reference github.com/colina118/Multin-cleo

__global__ void multiplyElements(int *a, int *b, int *c) {
  // Get the thread ID.
  int tid = threadIdx.x;

  // Calculate the element index.
  int index = blockIdx.x * blockDim.x + tid;

  // Multiply the elements and store the result.
  c[index] = a[index] * b[index];
}

int main() {
  // Declare the arrays on the host.
  int size = 10;
  int a[size], b[size], c[size];

  // Initialize the arrays.
  for (int i = 0; i < size; i++) {
    a[i] = i;
    b[i] = i * 2;
  }

  // Allocate memory on the device.
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, size * sizeof(int));
  hipMalloc(&d_b, size * sizeof(int));
  hipMalloc(&d_c, size * sizeof(int));

  // Copy the arrays to the device.
  hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

  // Launch the kernel.
  dim3 grid(1, 1); //grid size
  dim3 block(1024, 1); //block size: 1 is the dimension, each block has 1024 threads
  multiplyElements<<<grid, block>>>(d_a, d_b, d_c);

  // Copy the results back to the host.
  hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

  // Print the results.
  for (int i = 0; i < size; i++) {
    cout <<a[i]<<"x"<<b[i]<<"="<< c[i] << endl;
  }

  // Free the memory on the device.
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}

