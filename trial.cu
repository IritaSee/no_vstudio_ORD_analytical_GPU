#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(){
    printf("Hello Cuda\n");
}
 // compile like usual c code (nvcc trial.cu -o trial) will create trial.exe
int main(){
    //syntax:
    //[global or host void name]<<<kernel invocation, kernel threads>>>();
    dkernel<<<1,10>>>(); //these are called kernels -> when called, are executed 32 times in parallel by 32 different CUDA threads, as opposed to only once like regular C functions.
    /*
    A kernel is defined using the __global__ declaration specifier and the number of CUDA threads 
    that execute that kernel for a given kernel call is specified using a new <<<...>>> execution
    configuration syntax (see C Language Extensions). Each thread that executes the kernel is given a unique 
    thread ID that is accessible within the kernel through the built-in threadIdx variable.
    */
    hipDeviceSynchronize();
    /*
    The CUDA code you have provided is syntactically correct. 
    However, it will not print anything to the console. 
    This is because the kernel is not actually running on the GPU. 
    To run the kernel on the GPU, you need to call the cudaDeviceSynchronize() function. 
    This function will ensure that the kernel has finished executing before the main() function returns.
    */
    //cudaThreadSynchronize(); -> deprecated
    return 0;
}